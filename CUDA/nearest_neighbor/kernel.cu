#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <cstdio>
#include <iostream>
#include <IL/il.h>
#include "image.h"
#include "gputimer.h"

#define FACTOR 4

__global__ void nearest_neighbor(uint32_t *dp, uint32_t *sp, ILuint height, ILuint width, int factor)
{
	for (int row = 0; row < height; ++row) {
		for (int col = 0; col < width; ++col) {
			dp[(row * width + col)] = sp[((row / factor) * width/factor + (col / factor))];
		}
	}
}


int main()
{
	ILuint handle;
	ilInit();
	ilEnable(IL_ORIGIN_SET);
	ilGenImages(1, &handle);
	ilBindImage(handle);
	hipError_t cudaStatus;


	std::cout << "Read images" << std::endl;
	Image *org = new Image("image.png");
	Image *res = new Image("image2.png", org->getWidth() * FACTOR, org->getHeight() * FACTOR);

	std::cout << "Execute nearest neighbor algorithm" << std::endl;
	
	uint32_t *input, *out, *temp;
	
	// take first device
	cudaStatus = hipSetDevice(0);

	// alloc memory
	hipMalloc(&input, org->getHeight() * org->getWidth() * sizeof(uint32_t));
	hipMalloc(&out, res->getHeight() * res->getWidth() * sizeof(uint32_t));
	
	// copy original image to GPU
	hipMemcpy(input, org->getData(), org->getHeight() * org->getWidth() * sizeof(uint32_t), hipMemcpyHostToDevice);
	
	// Execute Kernel and measure time
	GpuTimer timer;

	timer.Start();
	nearest_neighbor<<<1,1>>>(out, input, res->getHeight(), res->getWidth(), 4);
	
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

	// wait that all kernels finish
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	}
	timer.Stop();

	std::cout << "Kernel time: " << timer.Elapsed() << std::endl;

	// copy new image to host
	cudaStatus = hipMemcpy(res->getData(), out, res->getHeight() * res->getWidth() * sizeof(uint32_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	std::cout << "Save new image" << std::endl;
	res->saveImage();

	std::cout << "Free space" << std::endl;
	delete org;
	delete res;
	ilDeleteImages(1, &handle);

	hipFree(input);
	hipFree(out);

    return 0;
}
