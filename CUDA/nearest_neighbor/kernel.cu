#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <cstdio>
#include <iostream>
#include <IL/il.h>
#include "image.h"
#include "gputimer.h"

#define FACTOR 4

__global__ void nearest_neighbor(uint32_t *dp, uint32_t *sp, ILuint width)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;

	int row = i / width;
	int col = i % width;
	dp[i] = sp[((row / FACTOR) * width/FACTOR + (col / FACTOR))];
}


int main()
{

	ILuint handle;
	ilInit();
	ilEnable(IL_ORIGIN_SET);
	ilGenImages(1, &handle);
	ilBindImage(handle);
	hipError_t cudaStatus;

	// get device properties
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);

	std::cout << "Read images" << std::endl;
	Image *org = new Image("image.png");
	Image *res = new Image("image2.png", org->getWidth() * FACTOR, org->getHeight() * FACTOR);

	std::cout << "Execute nearest neighbor algorithm" << std::endl;
	uint32_t *input, *out;
	
	// take first device
	cudaStatus = hipSetDevice(0);

	// alloc memory
	hipMalloc(&input, org->getHeight() * org->getWidth() * sizeof(uint32_t));
	hipMalloc(&out, res->getHeight() * res->getWidth() * sizeof(uint32_t));
	
	// copy original image to GPU
	hipMemcpy(input, org->getData(), org->getHeight() * org->getWidth() * sizeof(uint32_t), hipMemcpyHostToDevice);
	
	// calculate blockdimension and thread size
	int blockdimension = (res->getHeight() * res->getWidth()) / deviceProp.maxThreadsPerBlock;
	int threadsize = deviceProp.maxThreadsPerBlock;

	// Execute Kernel and measure time
	GpuTimer timer;
	timer.Start();
	nearest_neighbor<<<blockdimension, threadsize>>>(out, input, res->getWidth());
	
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

	// wait that all kernels finish
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	}
	timer.Stop();

	std::cout << "Kernel time: " << timer.Elapsed() << std::endl;

	// copy new image to host
	cudaStatus = hipMemcpy(res->getData(), out, res->getHeight() * res->getWidth() * sizeof(uint32_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	std::cout << "Save new image" << std::endl;
	res->saveImage();

	std::cout << "Free space" << std::endl;
	delete org;
	delete res;
	ilDeleteImages(1, &handle);

	hipFree(input);
	hipFree(out);

	std:getchar();

    return 0;
}
