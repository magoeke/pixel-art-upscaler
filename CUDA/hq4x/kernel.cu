#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdint.h>
#include <stdlib.h>
#include <IL/il.h>
#include <stdint.h>
#include "common.h"
#include "image.h"
#include <cstdio>
#include <iostream>
#include "gputimer.h"

#define FACTOR 4

#include <stdio.h>

uint32_t   RGBtoYUV[16777216];
uint32_t   YUV1, YUV2;

void hqxInit();
void hq4x_32(uint32_t * sp, uint32_t * dp, int Xres, int Yres, uint32_t*);
__global__ void hq4x(uint32_t*, uint32_t, uint32_t*, uint32_t, int, int, uint32_t*, int);
void convertToYUV(Image*, Image*);

int main()
{
	ILuint handle;
	ilInit();
	ilEnable(IL_ORIGIN_SET);
	ilGenImages(1, &handle);
	ilBindImage(handle);
	hipError_t cudaStatus;
	hipDeviceProp_t deviceProp;

	hipGetDeviceProperties(&deviceProp, 0);

	std::cout << "Read images" << std::endl;
	Image *org = new Image("image.png");
	Image *orgy = new Image("image.png");
	//uint32_t *input, *out;
	hqxInit();
	
	//input = (uint32_t*) malloc(org->getHeight() * org->getWidth() * sizeof(uint32_t));
	//out = (uint32_t*)malloc(org->getHeight() * org->getWidth() * sizeof(uint32_t));
	
	convertToYUV(orgy, org);
	Image *res = new Image("image2.png", org->getWidth() * FACTOR, org->getHeight() * FACTOR);

	
	hq4x_32(org->getData(), res->getData(), org->getWidth(), org->getHeight(), orgy->getData());

	std::cout << "Save new image" << std::endl;
	res->saveImage();

	std::cout << "Free space" << std::endl;
	delete org;
	delete res;
	ilDeleteImages(1, &handle);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	std:getchar();

    return 0;
}

void convertToYUV(Image *orgy,Image *org) {
	uint32_t *br = orgy->getData();
	uint32_t *sp = org->getData();

	for (int row = 0; row < org->getHeight() * org->getWidth(); ++row) {
			br[row] = rgb_to_yuv(sp[row]);
	}
	//return size;
}

void hqxInit(void)
{
	/* Initalize RGB to YUV lookup table */
	uint32_t c, r, g, b, y, u, v;
	for (c = 0; c < 16777215; c++) {
		r = (c & 0xFF0000) >> 16;
		g = (c & 0x00FF00) >> 8;
		b = c & 0x0000FF;
		y = (uint32_t)(0.299*r + 0.587*g + 0.114*b);
		u = (uint32_t)(-0.169*r - 0.331*g + 0.5*b) + 128;
		v = (uint32_t)(0.5*r - 0.419*g - 0.081*b) + 128;
		RGBtoYUV[c] = (y << 16) + (u << 8) + v;
	}
}

//Lookup ged�ns
#define PIXEL00_0     *(dp+indexVier) = w[5];
#define PIXEL00_11    *(dp+indexVier) = Interp1(w[5], w[4]);
#define PIXEL00_12    *(dp+indexVier) = Interp1(w[5], w[2]);
#define PIXEL00_20    *(dp+indexVier) = Interp2(w[5], w[2], w[4]);
#define PIXEL00_50    *(dp+indexVier) = Interp5(w[2], w[4]);
#define PIXEL00_80    *(dp+indexVier) = Interp8(w[5], w[1]);
#define PIXEL00_81    *(dp+indexVier) = Interp8(w[5], w[4]);
#define PIXEL00_82    *(dp+indexVier) = Interp8(w[5], w[2]);
#define PIXEL01_0     *(dp+indexVier+1) = w[5];
#define PIXEL01_10    *(dp+indexVier+1) = Interp1(w[5], w[1]);
#define PIXEL01_12    *(dp+indexVier+1) = Interp1(w[5], w[2]);
#define PIXEL01_14    *(dp+indexVier+1) = Interp1(w[2], w[5]);
#define PIXEL01_21    *(dp+indexVier+1) = Interp2(w[2], w[5], w[4]);
#define PIXEL01_31    *(dp+indexVier+1) = Interp3(w[5], w[4]);
#define PIXEL01_50    *(dp+indexVier+1) = Interp5(w[2], w[5]);
#define PIXEL01_60    *(dp+indexVier+1) = Interp6(w[5], w[2], w[4]);
#define PIXEL01_61    *(dp+indexVier+1) = Interp6(w[5], w[2], w[1]);
#define PIXEL01_82    *(dp+indexVier+1) = Interp8(w[5], w[2]);
#define PIXEL01_83    *(dp+indexVier+1) = Interp8(w[2], w[4]);
#define PIXEL02_0     *(dp+indexVier+2) = w[5];
#define PIXEL02_10    *(dp+indexVier+2) = Interp1(w[5], w[3]);
#define PIXEL02_11    *(dp+indexVier+2) = Interp1(w[5], w[2]);
#define PIXEL02_13    *(dp+indexVier+2) = Interp1(w[2], w[5]);
#define PIXEL02_21    *(dp+indexVier+2) = Interp2(w[2], w[5], w[6]);
#define PIXEL02_32    *(dp+indexVier+2) = Interp3(w[5], w[6]);
#define PIXEL02_50    *(dp+indexVier+2) = Interp5(w[2], w[5]);
#define PIXEL02_60    *(dp+indexVier+2) = Interp6(w[5], w[2], w[6]);
#define PIXEL02_61    *(dp+indexVier+2) = Interp6(w[5], w[2], w[3]);
#define PIXEL02_81    *(dp+indexVier+2) = Interp8(w[5], w[2]);
#define PIXEL02_83    *(dp+indexVier+2) = Interp8(w[2], w[6]);
#define PIXEL03_0     *(dp+indexVier+3) = w[5];
#define PIXEL03_11    *(dp+indexVier+3) = Interp1(w[5], w[2]);
#define PIXEL03_12    *(dp+indexVier+3) = Interp1(w[5], w[6]);
#define PIXEL03_20    *(dp+indexVier+3) = Interp2(w[5], w[2], w[6]);
#define PIXEL03_50    *(dp+indexVier+3) = Interp5(w[2], w[6]);
#define PIXEL03_80    *(dp+indexVier+3) = Interp8(w[5], w[3]);
#define PIXEL03_81    *(dp+indexVier+3) = Interp8(w[5], w[2]);
#define PIXEL03_82    *(dp+indexVier+3) = Interp8(w[5], w[6]);
#define PIXEL10_0     *(dp+indexVier+dpL) = w[5];
#define PIXEL10_10    *(dp+indexVier+dpL) = Interp1(w[5], w[1]);
#define PIXEL10_11    *(dp+indexVier+dpL) = Interp1(w[5], w[4]);
#define PIXEL10_13    *(dp+indexVier+dpL) = Interp1(w[4], w[5]);
#define PIXEL10_21    *(dp+indexVier+dpL) = Interp2(w[4], w[5], w[2]);
#define PIXEL10_32    *(dp+indexVier+dpL) = Interp3(w[5], w[2]);
#define PIXEL10_50    *(dp+indexVier+dpL) = Interp5(w[4], w[5]);
#define PIXEL10_60    *(dp+indexVier+dpL) = Interp6(w[5], w[4], w[2]);
#define PIXEL10_61    *(dp+indexVier+dpL) = Interp6(w[5], w[4], w[1]);
#define PIXEL10_81    *(dp+indexVier+dpL) = Interp8(w[5], w[4]);
#define PIXEL10_83    *(dp+indexVier+dpL) = Interp8(w[4], w[2]);
#define PIXEL11_0     *(dp+indexVier+dpL+1) = w[5];
#define PIXEL11_30    *(dp+indexVier+dpL+1) = Interp3(w[5], w[1]);
#define PIXEL11_31    *(dp+indexVier+dpL+1) = Interp3(w[5], w[4]);
#define PIXEL11_32    *(dp+indexVier+dpL+1) = Interp3(w[5], w[2]);
#define PIXEL11_70    *(dp+indexVier+dpL+1) = Interp7(w[5], w[4], w[2]);
#define PIXEL12_0     *(dp+indexVier+dpL+2) = w[5];
#define PIXEL12_30    *(dp+indexVier+dpL+2) = Interp3(w[5], w[3]);
#define PIXEL12_31    *(dp+indexVier+dpL+2) = Interp3(w[5], w[2]);
#define PIXEL12_32    *(dp+indexVier+dpL+2) = Interp3(w[5], w[6]);
#define PIXEL12_70    *(dp+indexVier+dpL+2) = Interp7(w[5], w[6], w[2]);
#define PIXEL13_0     *(dp+indexVier+dpL+3) = w[5];
#define PIXEL13_10    *(dp+indexVier+dpL+3) = Interp1(w[5], w[3]);
#define PIXEL13_12    *(dp+indexVier+dpL+3) = Interp1(w[5], w[6]);
#define PIXEL13_14    *(dp+indexVier+dpL+3) = Interp1(w[6], w[5]);
#define PIXEL13_21    *(dp+indexVier+dpL+3) = Interp2(w[6], w[5], w[2]);
#define PIXEL13_31    *(dp+indexVier+dpL+3) = Interp3(w[5], w[2]);
#define PIXEL13_50    *(dp+indexVier+dpL+3) = Interp5(w[6], w[5]);
#define PIXEL13_60    *(dp+indexVier+dpL+3) = Interp6(w[5], w[6], w[2]);
#define PIXEL13_61    *(dp+indexVier+dpL+3) = Interp6(w[5], w[6], w[3]);
#define PIXEL13_82    *(dp+indexVier+dpL+3) = Interp8(w[5], w[6]);
#define PIXEL13_83    *(dp+indexVier+dpL+3) = Interp8(w[6], w[2]);
#define PIXEL20_0     *(dp+indexVier+dpL+dpL) = w[5];
#define PIXEL20_10    *(dp+indexVier+dpL+dpL) = Interp1(w[5], w[7]);
#define PIXEL20_12    *(dp+indexVier+dpL+dpL) = Interp1(w[5], w[4]);
#define PIXEL20_14    *(dp+indexVier+dpL+dpL) = Interp1(w[4], w[5]);
#define PIXEL20_21    *(dp+indexVier+dpL+dpL) = Interp2(w[4], w[5], w[8]);
#define PIXEL20_31    *(dp+indexVier+dpL+dpL) = Interp3(w[5], w[8]);
#define PIXEL20_50    *(dp+indexVier+dpL+dpL) = Interp5(w[4], w[5]);
#define PIXEL20_60    *(dp+indexVier+dpL+dpL) = Interp6(w[5], w[4], w[8]);
#define PIXEL20_61    *(dp+indexVier+dpL+dpL) = Interp6(w[5], w[4], w[7]);
#define PIXEL20_82    *(dp+indexVier+dpL+dpL) = Interp8(w[5], w[4]);
#define PIXEL20_83    *(dp+indexVier+dpL+dpL) = Interp8(w[4], w[8]);
#define PIXEL21_0     *(dp+indexVier+dpL+dpL+1) = w[5];
#define PIXEL21_30    *(dp+indexVier+dpL+dpL+1) = Interp3(w[5], w[7]);
#define PIXEL21_31    *(dp+indexVier+dpL+dpL+1) = Interp3(w[5], w[8]);
#define PIXEL21_32    *(dp+indexVier+dpL+dpL+1) = Interp3(w[5], w[4]);
#define PIXEL21_70    *(dp+indexVier+dpL+dpL+1) = Interp7(w[5], w[4], w[8]);
#define PIXEL22_0     *(dp+indexVier+dpL+dpL+2) = w[5];
#define PIXEL22_30    *(dp+indexVier+dpL+dpL+2) = Interp3(w[5], w[9]);
#define PIXEL22_31    *(dp+indexVier+dpL+dpL+2) = Interp3(w[5], w[6]);
#define PIXEL22_32    *(dp+indexVier+dpL+dpL+2) = Interp3(w[5], w[8]);
#define PIXEL22_70    *(dp+indexVier+dpL+dpL+2) = Interp7(w[5], w[6], w[8]);
#define PIXEL23_0     *(dp+indexVier+dpL+dpL+3) = w[5];
#define PIXEL23_10    *(dp+indexVier+dpL+dpL+3) = Interp1(w[5], w[9]);
#define PIXEL23_11    *(dp+indexVier+dpL+dpL+3) = Interp1(w[5], w[6]);
#define PIXEL23_13    *(dp+indexVier+dpL+dpL+3) = Interp1(w[6], w[5]);
#define PIXEL23_21    *(dp+indexVier+dpL+dpL+3) = Interp2(w[6], w[5], w[8]);
#define PIXEL23_32    *(dp+indexVier+dpL+dpL+3) = Interp3(w[5], w[8]);
#define PIXEL23_50    *(dp+indexVier+dpL+dpL+3) = Interp5(w[6], w[5]);
#define PIXEL23_60    *(dp+indexVier+dpL+dpL+3) = Interp6(w[5], w[6], w[8]);
#define PIXEL23_61    *(dp+indexVier+dpL+dpL+3) = Interp6(w[5], w[6], w[9]);
#define PIXEL23_81    *(dp+indexVier+dpL+dpL+3) = Interp8(w[5], w[6]);
#define PIXEL23_83    *(dp+indexVier+dpL+dpL+3) = Interp8(w[6], w[8]);
#define PIXEL30_0     *(dp+indexVier+dpL+dpL+dpL) = w[5];
#define PIXEL30_11    *(dp+indexVier+dpL+dpL+dpL) = Interp1(w[5], w[8]);
#define PIXEL30_12    *(dp+indexVier+dpL+dpL+dpL) = Interp1(w[5], w[4]);
#define PIXEL30_20    *(dp+indexVier+dpL+dpL+dpL) = Interp2(w[5], w[8], w[4]);
#define PIXEL30_50    *(dp+indexVier+dpL+dpL+dpL) = Interp5(w[8], w[4]);
#define PIXEL30_80    *(dp+indexVier+dpL+dpL+dpL) = Interp8(w[5], w[7]);
#define PIXEL30_81    *(dp+indexVier+dpL+dpL+dpL) = Interp8(w[5], w[8]);
#define PIXEL30_82    *(dp+indexVier+dpL+dpL+dpL) = Interp8(w[5], w[4]);
#define PIXEL31_0     *(dp+indexVier+dpL+dpL+dpL+1) = w[5];
#define PIXEL31_10    *(dp+indexVier+dpL+dpL+dpL+1) = Interp1(w[5], w[7]);
#define PIXEL31_11    *(dp+indexVier+dpL+dpL+dpL+1) = Interp1(w[5], w[8]);
#define PIXEL31_13    *(dp+indexVier+dpL+dpL+dpL+1) = Interp1(w[8], w[5]);
#define PIXEL31_21    *(dp+indexVier+dpL+dpL+dpL+1) = Interp2(w[8], w[5], w[4]);
#define PIXEL31_32    *(dp+indexVier+dpL+dpL+dpL+1) = Interp3(w[5], w[4]);
#define PIXEL31_50    *(dp+indexVier+dpL+dpL+dpL+1) = Interp5(w[8], w[5]);
#define PIXEL31_60    *(dp+indexVier+dpL+dpL+dpL+1) = Interp6(w[5], w[8], w[4]);
#define PIXEL31_61    *(dp+indexVier+dpL+dpL+dpL+1) = Interp6(w[5], w[8], w[7]);
#define PIXEL31_81    *(dp+indexVier+dpL+dpL+dpL+1) = Interp8(w[5], w[8]);
#define PIXEL31_83    *(dp+indexVier+dpL+dpL+dpL+1) = Interp8(w[8], w[4]);
#define PIXEL32_0     *(dp+indexVier+dpL+dpL+dpL+2) = w[5];
#define PIXEL32_10    *(dp+indexVier+dpL+dpL+dpL+2) = Interp1(w[5], w[9]);
#define PIXEL32_12    *(dp+indexVier+dpL+dpL+dpL+2) = Interp1(w[5], w[8]);
#define PIXEL32_14    *(dp+indexVier+dpL+dpL+dpL+2) = Interp1(w[8], w[5]);
#define PIXEL32_21    *(dp+indexVier+dpL+dpL+dpL+2) = Interp2(w[8], w[5], w[6]);
#define PIXEL32_31    *(dp+indexVier+dpL+dpL+dpL+2) = Interp3(w[5], w[6]);
#define PIXEL32_50    *(dp+indexVier+dpL+dpL+dpL+2) = Interp5(w[8], w[5]);
#define PIXEL32_60    *(dp+indexVier+dpL+dpL+dpL+2) = Interp6(w[5], w[8], w[6]);
#define PIXEL32_61    *(dp+indexVier+dpL+dpL+dpL+2) = Interp6(w[5], w[8], w[9]);
#define PIXEL32_82    *(dp+indexVier+dpL+dpL+dpL+2) = Interp8(w[5], w[8]);
#define PIXEL32_83    *(dp+indexVier+dpL+dpL+dpL+2) = Interp8(w[8], w[6]);
#define PIXEL33_0     *(dp+indexVier+dpL+dpL+dpL+3) = w[5];
#define PIXEL33_11    *(dp+indexVier+dpL+dpL+dpL+3) = Interp1(w[5], w[6]);
#define PIXEL33_80    *(dp+indexVier+dpL+dpL+dpL+3) = Interp8(w[5], w[9]);
#define PIXEL33_12    *(dp+indexVier+dpL+dpL+dpL+3) = Interp1(w[5], w[8]);
#define PIXEL33_20    *(dp+indexVier+dpL+dpL+dpL+3) = Interp2(w[5], w[8], w[6]);
#define PIXEL33_50    *(dp+indexVier+dpL+dpL+dpL+3) = Interp5(w[8], w[6]);
#define PIXEL33_81    *(dp+indexVier+dpL+dpL+dpL+3) = Interp8(w[5], w[6]);
#define PIXEL33_82    *(dp+indexVier+dpL+dpL+dpL+3) = Interp8(w[5], w[8]);

void hq4x_32(uint32_t * sp, uint32_t * dp, int Xres, int Yres, uint32_t * orgy)
{
	hipError_t cudaStatus;
	uint32_t rowBytesL = Xres * FACTOR;
	uint32_t *input, *out, *yuv;
	hipDeviceProp_t deviceProp;

	hipGetDeviceProperties(&deviceProp, 0);

	cudaStatus = hipSetDevice(0);
	hipMalloc(&input, Xres * Yres * sizeof(uint32_t));
	hipMalloc(&out, Xres * FACTOR * Yres * FACTOR * sizeof(uint32_t));
	hipMalloc(&yuv, Xres * Yres * sizeof(uint32_t));

	hipMemcpy(input, sp, Xres * Yres * sizeof(uint32_t), hipMemcpyHostToDevice);
	hipMemcpy(yuv, orgy, Xres * Yres * sizeof(uint32_t), hipMemcpyHostToDevice);

	int threadSize = 256;

	int blockdimension = (Xres*Yres) > threadSize ? (Xres*Yres) / threadSize +1 : 1;

	GpuTimer timer;
	timer.Start();

	hq4x<<<blockdimension, threadSize>>>(input, rowBytesL, out, rowBytesL*FACTOR, Xres, Yres, yuv, threadSize);
	//hq4x_32_rb(sp, rowBytesL, dp, rowBytesL * 4, Xres, Yres);

	cudaStatus = hipDeviceSynchronize();
	timer.Stop();
	fprintf(stderr, "%g\n", timer.Elapsed());

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	}

	cudaStatus = hipMemcpy(dp, out,Xres *FACTOR * Yres * FACTOR * sizeof(uint32_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
}

uint32_t calcValuesArround(int prevline,int nexline) {
	
	return 0;
}

__global__ void hq4x(uint32_t * sp, uint32_t srb, uint32_t * dp, uint32_t drb, int Xres, int Yres, uint32_t *yuv, int maxThreads)
{
	int  k;
	int  prevline, nextline;
	uint32_t w[10];
	uint32_t y[10];
	uint32_t yuv1, yuv2;

	int index = threadIdx.x + maxThreads * blockIdx.x;

	int row = index / Xres;
	int col = index % Xres;

	int dpL = Xres*FACTOR;
	int spL = Xres;

	if (index < Xres) prevline = 0; else prevline = index - spL;
	if (index > Xres*(Yres - 2)) nextline = 0; else nextline = index + spL;

	int indexVier = (row * Xres * FACTOR + col) * FACTOR;


	w[2] = sp[prevline];
	w[5] = sp[index];
	w[8] = sp[nextline];
	y[2] = yuv[prevline];
	y[5] = yuv[index];
	y[8] = yuv[nextline];

	if ((index%Xres)>0)
	{
		w[1] = sp[prevline - 1];
		w[4] = sp[index - 1];
		w[7] = sp[nextline - 1];
		y[1] = yuv[prevline - 1];
		y[4] = yuv[index - 1];
		y[7] = yuv[nextline - 1];
	}
	else
	{
		w[1] = w[2];
		w[4] = w[5];
		w[7] = w[8];
		y[1] = y[2];
		y[4] = y[5];
		y[7] = y[8];
	}

	if ((index%Xres)<Xres - 1)
	{
		w[3] = sp[prevline + 1];
		w[6] = sp[index + 1];
		w[9] = sp[nextline + 1];
		y[3] = yuv[prevline + 1];
		y[6] = yuv[index + 1];
		y[9] = yuv[nextline + 1];
	}
	else
	{
		w[3] = w[2];
		w[6] = w[5];
		w[9] = w[8];
		y[3] = y[2];
		y[6] = y[5];
		y[9] = y[8];
	}

	int pattern = 0;
	int flag = 1;

	yuv1 = y[5];

	for (k = 1; k <= 9; k++)
	{
		if (k == 5) continue;

		if (w[k] != w[5])
		{
			yuv2 = y[k];
			if (yuv_diff(yuv1, yuv2))
				pattern |= flag;
		}
		flag <<= 1;
	}
	//dp[index + dpL + dpL + dpL + 2] = 8864886;
	
	switch (pattern)
	{
	case 0:
	case 1:
	case 4:
	case 32:
	case 128:
	case 5:
	case 132:
	case 160:
	case 33:
	case 129:
	case 36:
	case 133:
	case 164:
	case 161:
	case 37:
	case 165:
	{
		PIXEL00_20
			PIXEL01_60
			PIXEL02_60
			PIXEL03_20
			PIXEL10_60
			PIXEL11_70
			PIXEL12_70
			PIXEL13_60
			PIXEL20_60
			PIXEL21_70
			PIXEL22_70
			PIXEL23_60
			PIXEL30_20
			PIXEL31_60
			PIXEL32_60
			PIXEL33_20
			break;
	}
	case 2:
	case 34:
	case 130:
	case 162:
	{
		PIXEL00_80
			PIXEL01_10
			PIXEL02_10
			PIXEL03_80
			PIXEL10_61
			PIXEL11_30
			PIXEL12_30
			PIXEL13_61
			PIXEL20_60
			PIXEL21_70
			PIXEL22_70
			PIXEL23_60
			PIXEL30_20
			PIXEL31_60
			PIXEL32_60
			PIXEL33_20
			break;
	}
	case 16:
	case 17:
	case 48:
	case 49:
	{
		PIXEL00_20
			PIXEL01_60
			PIXEL02_61
			PIXEL03_80
			PIXEL10_60
			PIXEL11_70
			PIXEL12_30
			PIXEL13_10
			PIXEL20_60
			PIXEL21_70
			PIXEL22_30
			PIXEL23_10
			PIXEL30_20
			PIXEL31_60
			PIXEL32_61
			PIXEL33_80
			break;
	}
	case 64:
	case 65:
	case 68:
	case 69:
	{
		PIXEL00_20
			PIXEL01_60
			PIXEL02_60
			PIXEL03_20
			PIXEL10_60
			PIXEL11_70
			PIXEL12_70
			PIXEL13_60
			PIXEL20_61
			PIXEL21_30
			PIXEL22_30
			PIXEL23_61
			PIXEL30_80
			PIXEL31_10
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 8:
	case 12:
	case 136:
	case 140:
	{
		PIXEL00_80
			PIXEL01_61
			PIXEL02_60
			PIXEL03_20
			PIXEL10_10
			PIXEL11_30
			PIXEL12_70
			PIXEL13_60
			PIXEL20_10
			PIXEL21_30
			PIXEL22_70
			PIXEL23_60
			PIXEL30_80
			PIXEL31_61
			PIXEL32_60
			PIXEL33_20
			break;
	}
	case 3:
	case 35:
	case 131:
	case 163:
	{
		PIXEL00_81
			PIXEL01_31
			PIXEL02_10
			PIXEL03_80
			PIXEL10_81
			PIXEL11_31
			PIXEL12_30
			PIXEL13_61
			PIXEL20_60
			PIXEL21_70
			PIXEL22_70
			PIXEL23_60
			PIXEL30_20
			PIXEL31_60
			PIXEL32_60
			PIXEL33_20
			break;
	}
	case 6:
	case 38:
	case 134:
	case 166:
	{
		PIXEL00_80
			PIXEL01_10
			PIXEL02_32
			PIXEL03_82
			PIXEL10_61
			PIXEL11_30
			PIXEL12_32
			PIXEL13_82
			PIXEL20_60
			PIXEL21_70
			PIXEL22_70
			PIXEL23_60
			PIXEL30_20
			PIXEL31_60
			PIXEL32_60
			PIXEL33_20
			break;
	}
	case 20:
	case 21:
	case 52:
	case 53:
	{
		PIXEL00_20
			PIXEL01_60
			PIXEL02_81
			PIXEL03_81
			PIXEL10_60
			PIXEL11_70
			PIXEL12_31
			PIXEL13_31
			PIXEL20_60
			PIXEL21_70
			PIXEL22_30
			PIXEL23_10
			PIXEL30_20
			PIXEL31_60
			PIXEL32_61
			PIXEL33_80
			break;
	}
	case 144:
	case 145:
	case 176:
	case 177:
	{
		PIXEL00_20
			PIXEL01_60
			PIXEL02_61
			PIXEL03_80
			PIXEL10_60
			PIXEL11_70
			PIXEL12_30
			PIXEL13_10
			PIXEL20_60
			PIXEL21_70
			PIXEL22_32
			PIXEL23_32
			PIXEL30_20
			PIXEL31_60
			PIXEL32_82
			PIXEL33_82
			break;
	}
	case 192:
	case 193:
	case 196:
	case 197:
	{
		PIXEL00_20
			PIXEL01_60
			PIXEL02_60
			PIXEL03_20
			PIXEL10_60
			PIXEL11_70
			PIXEL12_70
			PIXEL13_60
			PIXEL20_61
			PIXEL21_30
			PIXEL22_31
			PIXEL23_81
			PIXEL30_80
			PIXEL31_10
			PIXEL32_31
			PIXEL33_81
			break;
	}
	case 96:
	case 97:
	case 100:
	case 101:
	{
		PIXEL00_20
			PIXEL01_60
			PIXEL02_60
			PIXEL03_20
			PIXEL10_60
			PIXEL11_70
			PIXEL12_70
			PIXEL13_60
			PIXEL20_82
			PIXEL21_32
			PIXEL22_30
			PIXEL23_61
			PIXEL30_82
			PIXEL31_32
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 40:
	case 44:
	case 168:
	case 172:
	{
		PIXEL00_80
			PIXEL01_61
			PIXEL02_60
			PIXEL03_20
			PIXEL10_10
			PIXEL11_30
			PIXEL12_70
			PIXEL13_60
			PIXEL20_31
			PIXEL21_31
			PIXEL22_70
			PIXEL23_60
			PIXEL30_81
			PIXEL31_81
			PIXEL32_60
			PIXEL33_20
			break;
	}
	case 9:
	case 13:
	case 137:
	case 141:
	{
		PIXEL00_82
			PIXEL01_82
			PIXEL02_60
			PIXEL03_20
			PIXEL10_32
			PIXEL11_32
			PIXEL12_70
			PIXEL13_60
			PIXEL20_10
			PIXEL21_30
			PIXEL22_70
			PIXEL23_60
			PIXEL30_80
			PIXEL31_61
			PIXEL32_60
			PIXEL33_20
			break;
	}
	case 18:
	case 50:
	{
		PIXEL00_80
			PIXEL01_10
			if (Diff(y[2], y[6]))
			{
				PIXEL02_10
					PIXEL03_80
					PIXEL12_30
					PIXEL13_10
			}
			else
			{
				PIXEL02_50
					PIXEL03_50
					PIXEL12_0
					PIXEL13_50
			}
		PIXEL10_61
			PIXEL11_30
			PIXEL20_60
			PIXEL21_70
			PIXEL22_30
			PIXEL23_10
			PIXEL30_20
			PIXEL31_60
			PIXEL32_61
			PIXEL33_80
			break;
	}
	case 80:
	case 81:
	{
		PIXEL00_20
			PIXEL01_60
			PIXEL02_61
			PIXEL03_80
			PIXEL10_60
			PIXEL11_70
			PIXEL12_30
			PIXEL13_10
			PIXEL20_61
			PIXEL21_30
			if (Diff(y[6], y[8]))
			{
				PIXEL22_30
					PIXEL23_10
					PIXEL32_10
					PIXEL33_80
			}
			else
			{
				PIXEL22_0
					PIXEL23_50
					PIXEL32_50
					PIXEL33_50
			}
		PIXEL30_80
			PIXEL31_10
			break;
	}
	case 72:
	case 76:
	{
		PIXEL00_80
			PIXEL01_61
			PIXEL02_60
			PIXEL03_20
			PIXEL10_10
			PIXEL11_30
			PIXEL12_70
			PIXEL13_60
			if (Diff(y[8], y[4]))
			{
				PIXEL20_10
					PIXEL21_30
					PIXEL30_80
					PIXEL31_10
			}
			else
			{
				PIXEL20_50
					PIXEL21_0
					PIXEL30_50
					PIXEL31_50
			}
		PIXEL22_30
			PIXEL23_61
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 10:
	case 138:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_80
				PIXEL01_10
				PIXEL10_10
				PIXEL11_30
		}
		else
		{
			PIXEL00_50
				PIXEL01_50
				PIXEL10_50
				PIXEL11_0
		}
		PIXEL02_10
			PIXEL03_80
			PIXEL12_30
			PIXEL13_61
			PIXEL20_10
			PIXEL21_30
			PIXEL22_70
			PIXEL23_60
			PIXEL30_80
			PIXEL31_61
			PIXEL32_60
			PIXEL33_20
			break;
	}
	case 66:
	{
		PIXEL00_80
			PIXEL01_10
			PIXEL02_10
			PIXEL03_80
			PIXEL10_61
			PIXEL11_30
			PIXEL12_30
			PIXEL13_61
			PIXEL20_61
			PIXEL21_30
			PIXEL22_30
			PIXEL23_61
			PIXEL30_80
			PIXEL31_10
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 24:
	{
		PIXEL00_80
			PIXEL01_61
			PIXEL02_61
			PIXEL03_80
			PIXEL10_10
			PIXEL11_30
			PIXEL12_30
			PIXEL13_10
			PIXEL20_10
			PIXEL21_30
			PIXEL22_30
			PIXEL23_10
			PIXEL30_80
			PIXEL31_61
			PIXEL32_61
			PIXEL33_80
			break;
	}
	case 7:
	case 39:
	case 135:
	{
		PIXEL00_81
			PIXEL01_31
			PIXEL02_32
			PIXEL03_82
			PIXEL10_81
			PIXEL11_31
			PIXEL12_32
			PIXEL13_82
			PIXEL20_60
			PIXEL21_70
			PIXEL22_70
			PIXEL23_60
			PIXEL30_20
			PIXEL31_60
			PIXEL32_60
			PIXEL33_20
			break;
	}
	case 148:
	case 149:
	case 180:
	{
		PIXEL00_20
			PIXEL01_60
			PIXEL02_81
			PIXEL03_81
			PIXEL10_60
			PIXEL11_70
			PIXEL12_31
			PIXEL13_31
			PIXEL20_60
			PIXEL21_70
			PIXEL22_32
			PIXEL23_32
			PIXEL30_20
			PIXEL31_60
			PIXEL32_82
			PIXEL33_82
			break;
	}
	case 224:
	case 228:
	case 225:
	{
		PIXEL00_20
			PIXEL01_60
			PIXEL02_60
			PIXEL03_20
			PIXEL10_60
			PIXEL11_70
			PIXEL12_70
			PIXEL13_60
			PIXEL20_82
			PIXEL21_32
			PIXEL22_31
			PIXEL23_81
			PIXEL30_82
			PIXEL31_32
			PIXEL32_31
			PIXEL33_81
			break;
	}
	case 41:
	case 169:
	case 45:
	{
		PIXEL00_82
			PIXEL01_82
			PIXEL02_60
			PIXEL03_20
			PIXEL10_32
			PIXEL11_32
			PIXEL12_70
			PIXEL13_60
			PIXEL20_31
			PIXEL21_31
			PIXEL22_70
			PIXEL23_60
			PIXEL30_81
			PIXEL31_81
			PIXEL32_60
			PIXEL33_20
			break;
	}
	case 22:
	case 54:
	{
		PIXEL00_80
			PIXEL01_10
			if (Diff(y[2], y[6]))
			{
				PIXEL02_0
					PIXEL03_0
					PIXEL13_0
			}
			else
			{
				PIXEL02_50
					PIXEL03_50
					PIXEL13_50
			}
		PIXEL10_61
			PIXEL11_30
			PIXEL12_0
			PIXEL20_60
			PIXEL21_70
			PIXEL22_30
			PIXEL23_10
			PIXEL30_20
			PIXEL31_60
			PIXEL32_61
			PIXEL33_80
			break;
	}
	case 208:
	case 209:
	{
		PIXEL00_20
			PIXEL01_60
			PIXEL02_61
			PIXEL03_80
			PIXEL10_60
			PIXEL11_70
			PIXEL12_30
			PIXEL13_10
			PIXEL20_61
			PIXEL21_30
			PIXEL22_0
			if (Diff(y[6], y[8]))
			{
				PIXEL23_0
					PIXEL32_0
					PIXEL33_0
			}
			else
			{
				PIXEL23_50
					PIXEL32_50
					PIXEL33_50
			}
		PIXEL30_80
			PIXEL31_10
			break;
	}
	case 104:
	case 108:
	{
		PIXEL00_80
			PIXEL01_61
			PIXEL02_60
			PIXEL03_20
			PIXEL10_10
			PIXEL11_30
			PIXEL12_70
			PIXEL13_60
			if (Diff(y[8], y[4]))
			{
				PIXEL20_0
					PIXEL30_0
					PIXEL31_0
			}
			else
			{
				PIXEL20_50
					PIXEL30_50
					PIXEL31_50
			}
		PIXEL21_0
			PIXEL22_30
			PIXEL23_61
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 11:
	case 139:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_0
				PIXEL01_0
				PIXEL10_0
		}
		else
		{
			PIXEL00_50
				PIXEL01_50
				PIXEL10_50
		}
		PIXEL02_10
			PIXEL03_80
			PIXEL11_0
			PIXEL12_30
			PIXEL13_61
			PIXEL20_10
			PIXEL21_30
			PIXEL22_70
			PIXEL23_60
			PIXEL30_80
			PIXEL31_61
			PIXEL32_60
			PIXEL33_20
			break;
	}
	case 19:
	case 51:
	{
		if (Diff(y[2], y[6]))
		{
			PIXEL00_81
				PIXEL01_31
				PIXEL02_10
				PIXEL03_80
				PIXEL12_30
				PIXEL13_10
		}
		else
		{
			PIXEL00_12
				PIXEL01_14
				PIXEL02_83
				PIXEL03_50
				PIXEL12_70
				PIXEL13_21
		}
		PIXEL10_81
			PIXEL11_31
			PIXEL20_60
			PIXEL21_70
			PIXEL22_30
			PIXEL23_10
			PIXEL30_20
			PIXEL31_60
			PIXEL32_61
			PIXEL33_80
			break;
	}
	case 146:
	case 178:
	{
		PIXEL00_80
			PIXEL01_10
			if (Diff(y[2], y[6]))
			{
				PIXEL02_10
					PIXEL03_80
					PIXEL12_30
					PIXEL13_10
					PIXEL23_32
					PIXEL33_82
			}
			else
			{
				PIXEL02_21
					PIXEL03_50
					PIXEL12_70
					PIXEL13_83
					PIXEL23_13
					PIXEL33_11
			}
		PIXEL10_61
			PIXEL11_30
			PIXEL20_60
			PIXEL21_70
			PIXEL22_32
			PIXEL30_20
			PIXEL31_60
			PIXEL32_82
			break;
	}
	case 84:
	case 85:
	{
		PIXEL00_20
			PIXEL01_60
			PIXEL02_81
			if (Diff(y[6], y[8]))
			{
				PIXEL03_81
					PIXEL13_31
					PIXEL22_30
					PIXEL23_10
					PIXEL32_10
					PIXEL33_80
			}
			else
			{
				PIXEL03_12
					PIXEL13_14
					PIXEL22_70
					PIXEL23_83
					PIXEL32_21
					PIXEL33_50
			}
		PIXEL10_60
			PIXEL11_70
			PIXEL12_31
			PIXEL20_61
			PIXEL21_30
			PIXEL30_80
			PIXEL31_10
			break;
	}
	case 112:
	case 113:
	{
		PIXEL00_20
			PIXEL01_60
			PIXEL02_61
			PIXEL03_80
			PIXEL10_60
			PIXEL11_70
			PIXEL12_30
			PIXEL13_10
			PIXEL20_82
			PIXEL21_32
			if (Diff(y[6], y[8]))
			{
				PIXEL22_30
					PIXEL23_10
					PIXEL30_82
					PIXEL31_32
					PIXEL32_10
					PIXEL33_80
			}
			else
			{
				PIXEL22_70
					PIXEL23_21
					PIXEL30_11
					PIXEL31_13
					PIXEL32_83
					PIXEL33_50
			}
		break;
	}
	case 200:
	case 204:
	{
		PIXEL00_80
			PIXEL01_61
			PIXEL02_60
			PIXEL03_20
			PIXEL10_10
			PIXEL11_30
			PIXEL12_70
			PIXEL13_60
			if (Diff(y[8], y[4]))
			{
				PIXEL20_10
					PIXEL21_30
					PIXEL30_80
					PIXEL31_10
					PIXEL32_31
					PIXEL33_81
			}
			else
			{
				PIXEL20_21
					PIXEL21_70
					PIXEL30_50
					PIXEL31_83
					PIXEL32_14
					PIXEL33_12
			}
		PIXEL22_31
			PIXEL23_81
			break;
	}
	case 73:
	case 77:
	{
		if (Diff(y[8], y[4]))
		{
			PIXEL00_82
				PIXEL10_32
				PIXEL20_10
				PIXEL21_30
				PIXEL30_80
				PIXEL31_10
		}
		else
		{
			PIXEL00_11
				PIXEL10_13
				PIXEL20_83
				PIXEL21_70
				PIXEL30_50
				PIXEL31_21
		}
		PIXEL01_82
			PIXEL02_60
			PIXEL03_20
			PIXEL11_32
			PIXEL12_70
			PIXEL13_60
			PIXEL22_30
			PIXEL23_61
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 42:
	case 170:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_80
				PIXEL01_10
				PIXEL10_10
				PIXEL11_30
				PIXEL20_31
				PIXEL30_81
		}
		else
		{
			PIXEL00_50
				PIXEL01_21
				PIXEL10_83
				PIXEL11_70
				PIXEL20_14
				PIXEL30_12
		}
		PIXEL02_10
			PIXEL03_80
			PIXEL12_30
			PIXEL13_61
			PIXEL21_31
			PIXEL22_70
			PIXEL23_60
			PIXEL31_81
			PIXEL32_60
			PIXEL33_20
			break;
	}
	case 14:
	case 142:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_80
				PIXEL01_10
				PIXEL02_32
				PIXEL03_82
				PIXEL10_10
				PIXEL11_30
		}
		else
		{
			PIXEL00_50
				PIXEL01_83
				PIXEL02_13
				PIXEL03_11
				PIXEL10_21
				PIXEL11_70
		}
		PIXEL12_32
			PIXEL13_82
			PIXEL20_10
			PIXEL21_30
			PIXEL22_70
			PIXEL23_60
			PIXEL30_80
			PIXEL31_61
			PIXEL32_60
			PIXEL33_20
			break;
	}
	case 67:
	{
		PIXEL00_81
			PIXEL01_31
			PIXEL02_10
			PIXEL03_80
			PIXEL10_81
			PIXEL11_31
			PIXEL12_30
			PIXEL13_61
			PIXEL20_61
			PIXEL21_30
			PIXEL22_30
			PIXEL23_61
			PIXEL30_80
			PIXEL31_10
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 70:
	{
		PIXEL00_80
			PIXEL01_10
			PIXEL02_32
			PIXEL03_82
			PIXEL10_61
			PIXEL11_30
			PIXEL12_32
			PIXEL13_82
			PIXEL20_61
			PIXEL21_30
			PIXEL22_30
			PIXEL23_61
			PIXEL30_80
			PIXEL31_10
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 28:
	{
		PIXEL00_80
			PIXEL01_61
			PIXEL02_81
			PIXEL03_81
			PIXEL10_10
			PIXEL11_30
			PIXEL12_31
			PIXEL13_31
			PIXEL20_10
			PIXEL21_30
			PIXEL22_30
			PIXEL23_10
			PIXEL30_80
			PIXEL31_61
			PIXEL32_61
			PIXEL33_80
			break;
	}
	case 152:
	{
		PIXEL00_80
			PIXEL01_61
			PIXEL02_61
			PIXEL03_80
			PIXEL10_10
			PIXEL11_30
			PIXEL12_30
			PIXEL13_10
			PIXEL20_10
			PIXEL21_30
			PIXEL22_32
			PIXEL23_32
			PIXEL30_80
			PIXEL31_61
			PIXEL32_82
			PIXEL33_82
			break;
	}
	case 194:
	{
		PIXEL00_80
			PIXEL01_10
			PIXEL02_10
			PIXEL03_80
			PIXEL10_61
			PIXEL11_30
			PIXEL12_30
			PIXEL13_61
			PIXEL20_61
			PIXEL21_30
			PIXEL22_31
			PIXEL23_81
			PIXEL30_80
			PIXEL31_10
			PIXEL32_31
			PIXEL33_81
			break;
	}
	case 98:
	{
		PIXEL00_80
			PIXEL01_10
			PIXEL02_10
			PIXEL03_80
			PIXEL10_61
			PIXEL11_30
			PIXEL12_30
			PIXEL13_61
			PIXEL20_82
			PIXEL21_32
			PIXEL22_30
			PIXEL23_61
			PIXEL30_82
			PIXEL31_32
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 56:
	{
		PIXEL00_80
			PIXEL01_61
			PIXEL02_61
			PIXEL03_80
			PIXEL10_10
			PIXEL11_30
			PIXEL12_30
			PIXEL13_10
			PIXEL20_31
			PIXEL21_31
			PIXEL22_30
			PIXEL23_10
			PIXEL30_81
			PIXEL31_81
			PIXEL32_61
			PIXEL33_80
			break;
	}
	case 25:
	{
		PIXEL00_82
			PIXEL01_82
			PIXEL02_61
			PIXEL03_80
			PIXEL10_32
			PIXEL11_32
			PIXEL12_30
			PIXEL13_10
			PIXEL20_10
			PIXEL21_30
			PIXEL22_30
			PIXEL23_10
			PIXEL30_80
			PIXEL31_61
			PIXEL32_61
			PIXEL33_80
			break;
	}
	case 26:
	case 31:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_0
				PIXEL01_0
				PIXEL10_0
		}
		else
		{
			PIXEL00_50
				PIXEL01_50
				PIXEL10_50
		}
		if (Diff(y[2], y[6]))
		{
			PIXEL02_0
				PIXEL03_0
				PIXEL13_0
		}
		else
		{
			PIXEL02_50
				PIXEL03_50
				PIXEL13_50
		}
		PIXEL11_0
			PIXEL12_0
			PIXEL20_10
			PIXEL21_30
			PIXEL22_30
			PIXEL23_10
			PIXEL30_80
			PIXEL31_61
			PIXEL32_61
			PIXEL33_80
			break;
	}
	case 82:
	case 214:
	{
		PIXEL00_80
			PIXEL01_10
			if (Diff(y[2], y[6]))
			{
				PIXEL02_0
					PIXEL03_0
					PIXEL13_0
			}
			else
			{
				PIXEL02_50
					PIXEL03_50
					PIXEL13_50
			}
		PIXEL10_61
			PIXEL11_30
			PIXEL12_0
			PIXEL20_61
			PIXEL21_30
			PIXEL22_0
			if (Diff(y[6], y[8]))
			{
				PIXEL23_0
					PIXEL32_0
					PIXEL33_0
			}
			else
			{
				PIXEL23_50
					PIXEL32_50
					PIXEL33_50
			}
		PIXEL30_80
			PIXEL31_10
			break;
	}
	case 88:
	case 248:
	{
		PIXEL00_80
			PIXEL01_61
			PIXEL02_61
			PIXEL03_80
			PIXEL10_10
			PIXEL11_30
			PIXEL12_30
			PIXEL13_10
			if (Diff(y[8], y[4]))
			{
				PIXEL20_0
					PIXEL30_0
					PIXEL31_0
			}
			else
			{
				PIXEL20_50
					PIXEL30_50
					PIXEL31_50
			}
		PIXEL21_0
			PIXEL22_0
			if (Diff(y[6], y[8]))
			{
				PIXEL23_0
					PIXEL32_0
					PIXEL33_0
			}
			else
			{
				PIXEL23_50
					PIXEL32_50
					PIXEL33_50
			}
		break;
	}
	case 74:
	case 107:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_0
				PIXEL01_0
				PIXEL10_0
		}
		else
		{
			PIXEL00_50
				PIXEL01_50
				PIXEL10_50
		}
		PIXEL02_10
			PIXEL03_80
			PIXEL11_0
			PIXEL12_30
			PIXEL13_61
			if (Diff(y[8], y[4]))
			{
				PIXEL20_0
					PIXEL30_0
					PIXEL31_0
			}
			else
			{
				PIXEL20_50
					PIXEL30_50
					PIXEL31_50
			}
		PIXEL21_0
			PIXEL22_30
			PIXEL23_61
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 27:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_0
				PIXEL01_0
				PIXEL10_0
		}
		else
		{
			PIXEL00_50
				PIXEL01_50
				PIXEL10_50
		}
		PIXEL02_10
			PIXEL03_80
			PIXEL11_0
			PIXEL12_30
			PIXEL13_10
			PIXEL20_10
			PIXEL21_30
			PIXEL22_30
			PIXEL23_10
			PIXEL30_80
			PIXEL31_61
			PIXEL32_61
			PIXEL33_80
			break;
	}
	case 86:
	{
		PIXEL00_80
			PIXEL01_10
			if (Diff(y[2], y[6]))
			{
				PIXEL02_0
					PIXEL03_0
					PIXEL13_0
			}
			else
			{
				PIXEL02_50
					PIXEL03_50
					PIXEL13_50
			}
		PIXEL10_61
			PIXEL11_30
			PIXEL12_0
			PIXEL20_61
			PIXEL21_30
			PIXEL22_30
			PIXEL23_10
			PIXEL30_80
			PIXEL31_10
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 216:
	{
		PIXEL00_80
			PIXEL01_61
			PIXEL02_61
			PIXEL03_80
			PIXEL10_10
			PIXEL11_30
			PIXEL12_30
			PIXEL13_10
			PIXEL20_10
			PIXEL21_30
			PIXEL22_0
			if (Diff(y[6], y[8]))
			{
				PIXEL23_0
					PIXEL32_0
					PIXEL33_0
			}
			else
			{
				PIXEL23_50
					PIXEL32_50
					PIXEL33_50
			}
		PIXEL30_80
			PIXEL31_10
			break;
	}
	case 106:
	{
		PIXEL00_80
			PIXEL01_10
			PIXEL02_10
			PIXEL03_80
			PIXEL10_10
			PIXEL11_30
			PIXEL12_30
			PIXEL13_61
			if (Diff(y[8], y[4]))
			{
				PIXEL20_0
					PIXEL30_0
					PIXEL31_0
			}
			else
			{
				PIXEL20_50
					PIXEL30_50
					PIXEL31_50
			}
		PIXEL21_0
			PIXEL22_30
			PIXEL23_61
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 30:
	{
		PIXEL00_80
			PIXEL01_10
			if (Diff(y[2], y[6]))
			{
				PIXEL02_0
					PIXEL03_0
					PIXEL13_0
			}
			else
			{
				PIXEL02_50
					PIXEL03_50
					PIXEL13_50
			}
		PIXEL10_10
			PIXEL11_30
			PIXEL12_0
			PIXEL20_10
			PIXEL21_30
			PIXEL22_30
			PIXEL23_10
			PIXEL30_80
			PIXEL31_61
			PIXEL32_61
			PIXEL33_80
			break;
	}
	case 210:
	{
		PIXEL00_80
			PIXEL01_10
			PIXEL02_10
			PIXEL03_80
			PIXEL10_61
			PIXEL11_30
			PIXEL12_30
			PIXEL13_10
			PIXEL20_61
			PIXEL21_30
			PIXEL22_0
			if (Diff(y[6], y[8]))
			{
				PIXEL23_0
					PIXEL32_0
					PIXEL33_0
			}
			else
			{
				PIXEL23_50
					PIXEL32_50
					PIXEL33_50
			}
		PIXEL30_80
			PIXEL31_10
			break;
	}
	case 120:
	{
		PIXEL00_80
			PIXEL01_61
			PIXEL02_61
			PIXEL03_80
			PIXEL10_10
			PIXEL11_30
			PIXEL12_30
			PIXEL13_10
			if (Diff(y[8], y[4]))
			{
				PIXEL20_0
					PIXEL30_0
					PIXEL31_0
			}
			else
			{
				PIXEL20_50
					PIXEL30_50
					PIXEL31_50
			}
		PIXEL21_0
			PIXEL22_30
			PIXEL23_10
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 75:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_0
				PIXEL01_0
				PIXEL10_0
		}
		else
		{
			PIXEL00_50
				PIXEL01_50
				PIXEL10_50
		}
		PIXEL02_10
			PIXEL03_80
			PIXEL11_0
			PIXEL12_30
			PIXEL13_61
			PIXEL20_10
			PIXEL21_30
			PIXEL22_30
			PIXEL23_61
			PIXEL30_80
			PIXEL31_10
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 29:
	{
		PIXEL00_82
			PIXEL01_82
			PIXEL02_81
			PIXEL03_81
			PIXEL10_32
			PIXEL11_32
			PIXEL12_31
			PIXEL13_31
			PIXEL20_10
			PIXEL21_30
			PIXEL22_30
			PIXEL23_10
			PIXEL30_80
			PIXEL31_61
			PIXEL32_61
			PIXEL33_80
			break;
	}
	case 198:
	{
		PIXEL00_80
			PIXEL01_10
			PIXEL02_32
			PIXEL03_82
			PIXEL10_61
			PIXEL11_30
			PIXEL12_32
			PIXEL13_82
			PIXEL20_61
			PIXEL21_30
			PIXEL22_31
			PIXEL23_81
			PIXEL30_80
			PIXEL31_10
			PIXEL32_31
			PIXEL33_81
			break;
	}
	case 184:
	{
		PIXEL00_80
			PIXEL01_61
			PIXEL02_61
			PIXEL03_80
			PIXEL10_10
			PIXEL11_30
			PIXEL12_30
			PIXEL13_10
			PIXEL20_31
			PIXEL21_31
			PIXEL22_32
			PIXEL23_32
			PIXEL30_81
			PIXEL31_81
			PIXEL32_82
			PIXEL33_82
			break;
	}
	case 99:
	{
		PIXEL00_81
			PIXEL01_31
			PIXEL02_10
			PIXEL03_80
			PIXEL10_81
			PIXEL11_31
			PIXEL12_30
			PIXEL13_61
			PIXEL20_82
			PIXEL21_32
			PIXEL22_30
			PIXEL23_61
			PIXEL30_82
			PIXEL31_32
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 57:
	{
		PIXEL00_82
			PIXEL01_82
			PIXEL02_61
			PIXEL03_80
			PIXEL10_32
			PIXEL11_32
			PIXEL12_30
			PIXEL13_10
			PIXEL20_31
			PIXEL21_31
			PIXEL22_30
			PIXEL23_10
			PIXEL30_81
			PIXEL31_81
			PIXEL32_61
			PIXEL33_80
			break;
	}
	case 71:
	{
		PIXEL00_81
			PIXEL01_31
			PIXEL02_32
			PIXEL03_82
			PIXEL10_81
			PIXEL11_31
			PIXEL12_32
			PIXEL13_82
			PIXEL20_61
			PIXEL21_30
			PIXEL22_30
			PIXEL23_61
			PIXEL30_80
			PIXEL31_10
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 156:
	{
		PIXEL00_80
			PIXEL01_61
			PIXEL02_81
			PIXEL03_81
			PIXEL10_10
			PIXEL11_30
			PIXEL12_31
			PIXEL13_31
			PIXEL20_10
			PIXEL21_30
			PIXEL22_32
			PIXEL23_32
			PIXEL30_80
			PIXEL31_61
			PIXEL32_82
			PIXEL33_82
			break;
	}
	case 226:
	{
		PIXEL00_80
			PIXEL01_10
			PIXEL02_10
			PIXEL03_80
			PIXEL10_61
			PIXEL11_30
			PIXEL12_30
			PIXEL13_61
			PIXEL20_82
			PIXEL21_32
			PIXEL22_31
			PIXEL23_81
			PIXEL30_82
			PIXEL31_32
			PIXEL32_31
			PIXEL33_81
			break;
	}
	case 60:
	{
		PIXEL00_80
			PIXEL01_61
			PIXEL02_81
			PIXEL03_81
			PIXEL10_10
			PIXEL11_30
			PIXEL12_31
			PIXEL13_31
			PIXEL20_31
			PIXEL21_31
			PIXEL22_30
			PIXEL23_10
			PIXEL30_81
			PIXEL31_81
			PIXEL32_61
			PIXEL33_80
			break;
	}
	case 195:
	{
		PIXEL00_81
			PIXEL01_31
			PIXEL02_10
			PIXEL03_80
			PIXEL10_81
			PIXEL11_31
			PIXEL12_30
			PIXEL13_61
			PIXEL20_61
			PIXEL21_30
			PIXEL22_31
			PIXEL23_81
			PIXEL30_80
			PIXEL31_10
			PIXEL32_31
			PIXEL33_81
			break;
	}
	case 102:
	{
		PIXEL00_80
			PIXEL01_10
			PIXEL02_32
			PIXEL03_82
			PIXEL10_61
			PIXEL11_30
			PIXEL12_32
			PIXEL13_82
			PIXEL20_82
			PIXEL21_32
			PIXEL22_30
			PIXEL23_61
			PIXEL30_82
			PIXEL31_32
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 153:
	{
		PIXEL00_82
			PIXEL01_82
			PIXEL02_61
			PIXEL03_80
			PIXEL10_32
			PIXEL11_32
			PIXEL12_30
			PIXEL13_10
			PIXEL20_10
			PIXEL21_30
			PIXEL22_32
			PIXEL23_32
			PIXEL30_80
			PIXEL31_61
			PIXEL32_82
			PIXEL33_82
			break;
	}
	case 58:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_80
				PIXEL01_10
				PIXEL10_10
				PIXEL11_30
		}
		else
		{
			PIXEL00_20
				PIXEL01_12
				PIXEL10_11
				PIXEL11_0
		}
		if (Diff(y[2], y[6]))
		{
			PIXEL02_10
				PIXEL03_80
				PIXEL12_30
				PIXEL13_10
		}
		else
		{
			PIXEL02_11
				PIXEL03_20
				PIXEL12_0
				PIXEL13_12
		}
		PIXEL20_31
			PIXEL21_31
			PIXEL22_30
			PIXEL23_10
			PIXEL30_81
			PIXEL31_81
			PIXEL32_61
			PIXEL33_80
			break;
	}
	case 83:
	{
		PIXEL00_81
			PIXEL01_31
			if (Diff(y[2], y[6]))
			{
				PIXEL02_10
					PIXEL03_80
					PIXEL12_30
					PIXEL13_10
			}
			else
			{
				PIXEL02_11
					PIXEL03_20
					PIXEL12_0
					PIXEL13_12
			}
		PIXEL10_81
			PIXEL11_31
			PIXEL20_61
			PIXEL21_30
			if (Diff(y[6], y[8]))
			{
				PIXEL22_30
					PIXEL23_10
					PIXEL32_10
					PIXEL33_80
			}
			else
			{
				PIXEL22_0
					PIXEL23_11
					PIXEL32_12
					PIXEL33_20
			}
		PIXEL30_80
			PIXEL31_10
			break;
	}
	case 92:
	{
		PIXEL00_80
			PIXEL01_61
			PIXEL02_81
			PIXEL03_81
			PIXEL10_10
			PIXEL11_30
			PIXEL12_31
			PIXEL13_31
			if (Diff(y[8], y[4]))
			{
				PIXEL20_10
					PIXEL21_30
					PIXEL30_80
					PIXEL31_10
			}
			else
			{
				PIXEL20_12
					PIXEL21_0
					PIXEL30_20
					PIXEL31_11
			}
		if (Diff(y[6], y[8]))
		{
			PIXEL22_30
				PIXEL23_10
				PIXEL32_10
				PIXEL33_80
		}
		else
		{
			PIXEL22_0
				PIXEL23_11
				PIXEL32_12
				PIXEL33_20
		}
		break;
	}
	case 202:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_80
				PIXEL01_10
				PIXEL10_10
				PIXEL11_30
		}
		else
		{
			PIXEL00_20
				PIXEL01_12
				PIXEL10_11
				PIXEL11_0
		}
		PIXEL02_10
			PIXEL03_80
			PIXEL12_30
			PIXEL13_61
			if (Diff(y[8], y[4]))
			{
				PIXEL20_10
					PIXEL21_30
					PIXEL30_80
					PIXEL31_10
			}
			else
			{
				PIXEL20_12
					PIXEL21_0
					PIXEL30_20
					PIXEL31_11
			}
		PIXEL22_31
			PIXEL23_81
			PIXEL32_31
			PIXEL33_81
			break;
	}
	case 78:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_80
				PIXEL01_10
				PIXEL10_10
				PIXEL11_30
		}
		else
		{
			PIXEL00_20
				PIXEL01_12
				PIXEL10_11
				PIXEL11_0
		}
		PIXEL02_32
			PIXEL03_82
			PIXEL12_32
			PIXEL13_82
			if (Diff(y[8], y[4]))
			{
				PIXEL20_10
					PIXEL21_30
					PIXEL30_80
					PIXEL31_10
			}
			else
			{
				PIXEL20_12
					PIXEL21_0
					PIXEL30_20
					PIXEL31_11
			}
		PIXEL22_30
			PIXEL23_61
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 154:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_80
				PIXEL01_10
				PIXEL10_10
				PIXEL11_30
		}
		else
		{
			PIXEL00_20
				PIXEL01_12
				PIXEL10_11
				PIXEL11_0
		}
		if (Diff(y[2], y[6]))
		{
			PIXEL02_10
				PIXEL03_80
				PIXEL12_30
				PIXEL13_10
		}
		else
		{
			PIXEL02_11
				PIXEL03_20
				PIXEL12_0
				PIXEL13_12
		}
		PIXEL20_10
			PIXEL21_30
			PIXEL22_32
			PIXEL23_32
			PIXEL30_80
			PIXEL31_61
			PIXEL32_82
			PIXEL33_82
			break;
	}
	case 114:
	{
		PIXEL00_80
			PIXEL01_10
			if (Diff(y[2], y[6]))
			{
				PIXEL02_10
					PIXEL03_80
					PIXEL12_30
					PIXEL13_10
			}
			else
			{
				PIXEL02_11
					PIXEL03_20
					PIXEL12_0
					PIXEL13_12
			}
		PIXEL10_61
			PIXEL11_30
			PIXEL20_82
			PIXEL21_32
			if (Diff(y[6], y[8]))
			{
				PIXEL22_30
					PIXEL23_10
					PIXEL32_10
					PIXEL33_80
			}
			else
			{
				PIXEL22_0
					PIXEL23_11
					PIXEL32_12
					PIXEL33_20
			}
		PIXEL30_82
			PIXEL31_32
			break;
	}
	case 89:
	{
		PIXEL00_82
			PIXEL01_82
			PIXEL02_61
			PIXEL03_80
			PIXEL10_32
			PIXEL11_32
			PIXEL12_30
			PIXEL13_10
			if (Diff(y[8], y[4]))
			{
				PIXEL20_10
					PIXEL21_30
					PIXEL30_80
					PIXEL31_10
			}
			else
			{
				PIXEL20_12
					PIXEL21_0
					PIXEL30_20
					PIXEL31_11
			}
		if (Diff(y[6], y[8]))
		{
			PIXEL22_30
				PIXEL23_10
				PIXEL32_10
				PIXEL33_80
		}
		else
		{
			PIXEL22_0
				PIXEL23_11
				PIXEL32_12
				PIXEL33_20
		}
		break;
	}
	case 90:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_80
				PIXEL01_10
				PIXEL10_10
				PIXEL11_30
		}
		else
		{
			PIXEL00_20
				PIXEL01_12
				PIXEL10_11
				PIXEL11_0
		}
		if (Diff(y[2], y[6]))
		{
			PIXEL02_10
				PIXEL03_80
				PIXEL12_30
				PIXEL13_10
		}
		else
		{
			PIXEL02_11
				PIXEL03_20
				PIXEL12_0
				PIXEL13_12
		}
		if (Diff(y[8], y[4]))
		{
			PIXEL20_10
				PIXEL21_30
				PIXEL30_80
				PIXEL31_10
		}
		else
		{
			PIXEL20_12
				PIXEL21_0
				PIXEL30_20
				PIXEL31_11
		}
		if (Diff(y[6], y[8]))
		{
			PIXEL22_30
				PIXEL23_10
				PIXEL32_10
				PIXEL33_80
		}
		else
		{
			PIXEL22_0
				PIXEL23_11
				PIXEL32_12
				PIXEL33_20
		}
		break;
	}
	case 55:
	case 23:
	{
		if (Diff(y[2], y[6]))
		{
			PIXEL00_81
				PIXEL01_31
				PIXEL02_0
				PIXEL03_0
				PIXEL12_0
				PIXEL13_0
		}
		else
		{
			PIXEL00_12
				PIXEL01_14
				PIXEL02_83
				PIXEL03_50
				PIXEL12_70
				PIXEL13_21
		}
		PIXEL10_81
			PIXEL11_31
			PIXEL20_60
			PIXEL21_70
			PIXEL22_30
			PIXEL23_10
			PIXEL30_20
			PIXEL31_60
			PIXEL32_61
			PIXEL33_80
			break;
	}
	case 182:
	case 150:
	{
		PIXEL00_80
			PIXEL01_10
			if (Diff(y[2], y[6]))
			{
				PIXEL02_0
					PIXEL03_0
					PIXEL12_0
					PIXEL13_0
					PIXEL23_32
					PIXEL33_82
			}
			else
			{
				PIXEL02_21
					PIXEL03_50
					PIXEL12_70
					PIXEL13_83
					PIXEL23_13
					PIXEL33_11
			}
		PIXEL10_61
			PIXEL11_30
			PIXEL20_60
			PIXEL21_70
			PIXEL22_32
			PIXEL30_20
			PIXEL31_60
			PIXEL32_82
			break;
	}
	case 213:
	case 212:
	{
		PIXEL00_20
			PIXEL01_60
			PIXEL02_81
			if (Diff(y[6], y[8]))
			{
				PIXEL03_81
					PIXEL13_31
					PIXEL22_0
					PIXEL23_0
					PIXEL32_0
					PIXEL33_0
			}
			else
			{
				PIXEL03_12
					PIXEL13_14
					PIXEL22_70
					PIXEL23_83
					PIXEL32_21
					PIXEL33_50
			}
		PIXEL10_60
			PIXEL11_70
			PIXEL12_31
			PIXEL20_61
			PIXEL21_30
			PIXEL30_80
			PIXEL31_10
			break;
	}
	case 241:
	case 240:
	{
		PIXEL00_20
			PIXEL01_60
			PIXEL02_61
			PIXEL03_80
			PIXEL10_60
			PIXEL11_70
			PIXEL12_30
			PIXEL13_10
			PIXEL20_82
			PIXEL21_32
			if (Diff(y[6], y[8]))
			{
				PIXEL22_0
					PIXEL23_0
					PIXEL30_82
					PIXEL31_32
					PIXEL32_0
					PIXEL33_0
			}
			else
			{
				PIXEL22_70
					PIXEL23_21
					PIXEL30_11
					PIXEL31_13
					PIXEL32_83
					PIXEL33_50
			}
		break;
	}
	case 236:
	case 232:
	{
		PIXEL00_80
			PIXEL01_61
			PIXEL02_60
			PIXEL03_20
			PIXEL10_10
			PIXEL11_30
			PIXEL12_70
			PIXEL13_60
			if (Diff(y[8], y[4]))
			{
				PIXEL20_0
					PIXEL21_0
					PIXEL30_0
					PIXEL31_0
					PIXEL32_31
					PIXEL33_81
			}
			else
			{
				PIXEL20_21
					PIXEL21_70
					PIXEL30_50
					PIXEL31_83
					PIXEL32_14
					PIXEL33_12
			}
		PIXEL22_31
			PIXEL23_81
			break;
	}
	case 109:
	case 105:
	{
		if (Diff(y[8], y[4]))
		{
			PIXEL00_82
				PIXEL10_32
				PIXEL20_0
				PIXEL21_0
				PIXEL30_0
				PIXEL31_0
		}
		else
		{
			PIXEL00_11
				PIXEL10_13
				PIXEL20_83
				PIXEL21_70
				PIXEL30_50
				PIXEL31_21
		}
		PIXEL01_82
			PIXEL02_60
			PIXEL03_20
			PIXEL11_32
			PIXEL12_70
			PIXEL13_60
			PIXEL22_30
			PIXEL23_61
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 171:
	case 43:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_0
				PIXEL01_0
				PIXEL10_0
				PIXEL11_0
				PIXEL20_31
				PIXEL30_81
		}
		else
		{
			PIXEL00_50
				PIXEL01_21
				PIXEL10_83
				PIXEL11_70
				PIXEL20_14
				PIXEL30_12
		}
		PIXEL02_10
			PIXEL03_80
			PIXEL12_30
			PIXEL13_61
			PIXEL21_31
			PIXEL22_70
			PIXEL23_60
			PIXEL31_81
			PIXEL32_60
			PIXEL33_20
			break;
	}
	case 143:
	case 15:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_0
				PIXEL01_0
				PIXEL02_32
				PIXEL03_82
				PIXEL10_0
				PIXEL11_0
		}
		else
		{
			PIXEL00_50
				PIXEL01_83
				PIXEL02_13
				PIXEL03_11
				PIXEL10_21
				PIXEL11_70
		}
		PIXEL12_32
			PIXEL13_82
			PIXEL20_10
			PIXEL21_30
			PIXEL22_70
			PIXEL23_60
			PIXEL30_80
			PIXEL31_61
			PIXEL32_60
			PIXEL33_20
			break;
	}
	case 124:
	{
		PIXEL00_80
			PIXEL01_61
			PIXEL02_81
			PIXEL03_81
			PIXEL10_10
			PIXEL11_30
			PIXEL12_31
			PIXEL13_31
			if (Diff(y[8], y[4]))
			{
				PIXEL20_0
					PIXEL30_0
					PIXEL31_0
			}
			else
			{
				PIXEL20_50
					PIXEL30_50
					PIXEL31_50
			}
		PIXEL21_0
			PIXEL22_30
			PIXEL23_10
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 203:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_0
				PIXEL01_0
				PIXEL10_0
		}
		else
		{
			PIXEL00_50
				PIXEL01_50
				PIXEL10_50
		}
		PIXEL02_10
			PIXEL03_80
			PIXEL11_0
			PIXEL12_30
			PIXEL13_61
			PIXEL20_10
			PIXEL21_30
			PIXEL22_31
			PIXEL23_81
			PIXEL30_80
			PIXEL31_10
			PIXEL32_31
			PIXEL33_81
			break;
	}
	case 62:
	{
		PIXEL00_80
			PIXEL01_10
			if (Diff(y[2], y[6]))
			{
				PIXEL02_0
					PIXEL03_0
					PIXEL13_0
			}
			else
			{
				PIXEL02_50
					PIXEL03_50
					PIXEL13_50
			}
		PIXEL10_10
			PIXEL11_30
			PIXEL12_0
			PIXEL20_31
			PIXEL21_31
			PIXEL22_30
			PIXEL23_10
			PIXEL30_81
			PIXEL31_81
			PIXEL32_61
			PIXEL33_80
			break;
	}
	case 211:
	{
		PIXEL00_81
			PIXEL01_31
			PIXEL02_10
			PIXEL03_80
			PIXEL10_81
			PIXEL11_31
			PIXEL12_30
			PIXEL13_10
			PIXEL20_61
			PIXEL21_30
			PIXEL22_0
			if (Diff(y[6], y[8]))
			{
				PIXEL23_0
					PIXEL32_0
					PIXEL33_0
			}
			else
			{
				PIXEL23_50
					PIXEL32_50
					PIXEL33_50
			}
		PIXEL30_80
			PIXEL31_10
			break;
	}
	case 118:
	{
		PIXEL00_80
			PIXEL01_10
			if (Diff(y[2], y[6]))
			{
				PIXEL02_0
					PIXEL03_0
					PIXEL13_0
			}
			else
			{
				PIXEL02_50
					PIXEL03_50
					PIXEL13_50
			}
		PIXEL10_61
			PIXEL11_30
			PIXEL12_0
			PIXEL20_82
			PIXEL21_32
			PIXEL22_30
			PIXEL23_10
			PIXEL30_82
			PIXEL31_32
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 217:
	{
		PIXEL00_82
			PIXEL01_82
			PIXEL02_61
			PIXEL03_80
			PIXEL10_32
			PIXEL11_32
			PIXEL12_30
			PIXEL13_10
			PIXEL20_10
			PIXEL21_30
			PIXEL22_0
			if (Diff(y[6], y[8]))
			{
				PIXEL23_0
					PIXEL32_0
					PIXEL33_0
			}
			else
			{
				PIXEL23_50
					PIXEL32_50
					PIXEL33_50
			}
		PIXEL30_80
			PIXEL31_10
			break;
	}
	case 110:
	{
		PIXEL00_80
			PIXEL01_10
			PIXEL02_32
			PIXEL03_82
			PIXEL10_10
			PIXEL11_30
			PIXEL12_32
			PIXEL13_82
			if (Diff(y[8], y[4]))
			{
				PIXEL20_0
					PIXEL30_0
					PIXEL31_0
			}
			else
			{
				PIXEL20_50
					PIXEL30_50
					PIXEL31_50
			}
		PIXEL21_0
			PIXEL22_30
			PIXEL23_61
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 155:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_0
				PIXEL01_0
				PIXEL10_0
		}
		else
		{
			PIXEL00_50
				PIXEL01_50
				PIXEL10_50
		}
		PIXEL02_10
			PIXEL03_80
			PIXEL11_0
			PIXEL12_30
			PIXEL13_10
			PIXEL20_10
			PIXEL21_30
			PIXEL22_32
			PIXEL23_32
			PIXEL30_80
			PIXEL31_61
			PIXEL32_82
			PIXEL33_82
			break;
	}
	case 188:
	{
		PIXEL00_80
			PIXEL01_61
			PIXEL02_81
			PIXEL03_81
			PIXEL10_10
			PIXEL11_30
			PIXEL12_31
			PIXEL13_31
			PIXEL20_31
			PIXEL21_31
			PIXEL22_32
			PIXEL23_32
			PIXEL30_81
			PIXEL31_81
			PIXEL32_82
			PIXEL33_82
			break;
	}
	case 185:
	{
		PIXEL00_82
			PIXEL01_82
			PIXEL02_61
			PIXEL03_80
			PIXEL10_32
			PIXEL11_32
			PIXEL12_30
			PIXEL13_10
			PIXEL20_31
			PIXEL21_31
			PIXEL22_32
			PIXEL23_32
			PIXEL30_81
			PIXEL31_81
			PIXEL32_82
			PIXEL33_82
			break;
	}
	case 61:
	{
		PIXEL00_82
			PIXEL01_82
			PIXEL02_81
			PIXEL03_81
			PIXEL10_32
			PIXEL11_32
			PIXEL12_31
			PIXEL13_31
			PIXEL20_31
			PIXEL21_31
			PIXEL22_30
			PIXEL23_10
			PIXEL30_81
			PIXEL31_81
			PIXEL32_61
			PIXEL33_80
			break;
	}
	case 157:
	{
		PIXEL00_82
			PIXEL01_82
			PIXEL02_81
			PIXEL03_81
			PIXEL10_32
			PIXEL11_32
			PIXEL12_31
			PIXEL13_31
			PIXEL20_10
			PIXEL21_30
			PIXEL22_32
			PIXEL23_32
			PIXEL30_80
			PIXEL31_61
			PIXEL32_82
			PIXEL33_82
			break;
	}
	case 103:
	{
		PIXEL00_81
			PIXEL01_31
			PIXEL02_32
			PIXEL03_82
			PIXEL10_81
			PIXEL11_31
			PIXEL12_32
			PIXEL13_82
			PIXEL20_82
			PIXEL21_32
			PIXEL22_30
			PIXEL23_61
			PIXEL30_82
			PIXEL31_32
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 227:
	{
		PIXEL00_81
			PIXEL01_31
			PIXEL02_10
			PIXEL03_80
			PIXEL10_81
			PIXEL11_31
			PIXEL12_30
			PIXEL13_61
			PIXEL20_82
			PIXEL21_32
			PIXEL22_31
			PIXEL23_81
			PIXEL30_82
			PIXEL31_32
			PIXEL32_31
			PIXEL33_81
			break;
	}
	case 230:
	{
		PIXEL00_80
			PIXEL01_10
			PIXEL02_32
			PIXEL03_82
			PIXEL10_61
			PIXEL11_30
			PIXEL12_32
			PIXEL13_82
			PIXEL20_82
			PIXEL21_32
			PIXEL22_31
			PIXEL23_81
			PIXEL30_82
			PIXEL31_32
			PIXEL32_31
			PIXEL33_81
			break;
	}
	case 199:
	{
		PIXEL00_81
			PIXEL01_31
			PIXEL02_32
			PIXEL03_82
			PIXEL10_81
			PIXEL11_31
			PIXEL12_32
			PIXEL13_82
			PIXEL20_61
			PIXEL21_30
			PIXEL22_31
			PIXEL23_81
			PIXEL30_80
			PIXEL31_10
			PIXEL32_31
			PIXEL33_81
			break;
	}
	case 220:
	{
		PIXEL00_80
			PIXEL01_61
			PIXEL02_81
			PIXEL03_81
			PIXEL10_10
			PIXEL11_30
			PIXEL12_31
			PIXEL13_31
			if (Diff(y[8], y[4]))
			{
				PIXEL20_10
					PIXEL21_30
					PIXEL30_80
					PIXEL31_10
			}
			else
			{
				PIXEL20_12
					PIXEL21_0
					PIXEL30_20
					PIXEL31_11
			}
		PIXEL22_0
			if (Diff(y[6], y[8]))
			{
				PIXEL23_0
					PIXEL32_0
					PIXEL33_0
			}
			else
			{
				PIXEL23_50
					PIXEL32_50
					PIXEL33_50
			}
		break;
	}
	case 158:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_80
				PIXEL01_10
				PIXEL10_10
				PIXEL11_30
		}
		else
		{
			PIXEL00_20
				PIXEL01_12
				PIXEL10_11
				PIXEL11_0
		}
		if (Diff(y[2], y[6]))
		{
			PIXEL02_0
				PIXEL03_0
				PIXEL13_0
		}
		else
		{
			PIXEL02_50
				PIXEL03_50
				PIXEL13_50
		}
		PIXEL12_0
			PIXEL20_10
			PIXEL21_30
			PIXEL22_32
			PIXEL23_32
			PIXEL30_80
			PIXEL31_61
			PIXEL32_82
			PIXEL33_82
			break;
	}
	case 234:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_80
				PIXEL01_10
				PIXEL10_10
				PIXEL11_30
		}
		else
		{
			PIXEL00_20
				PIXEL01_12
				PIXEL10_11
				PIXEL11_0
		}
		PIXEL02_10
			PIXEL03_80
			PIXEL12_30
			PIXEL13_61
			if (Diff(y[8], y[4]))
			{
				PIXEL20_0
					PIXEL30_0
					PIXEL31_0
			}
			else
			{
				PIXEL20_50
					PIXEL30_50
					PIXEL31_50
			}
		PIXEL21_0
			PIXEL22_31
			PIXEL23_81
			PIXEL32_31
			PIXEL33_81
			break;
	}
	case 242:
	{
		PIXEL00_80
			PIXEL01_10
			if (Diff(y[2], y[6]))
			{
				PIXEL02_10
					PIXEL03_80
					PIXEL12_30
					PIXEL13_10
			}
			else
			{
				PIXEL02_11
					PIXEL03_20
					PIXEL12_0
					PIXEL13_12
			}
		PIXEL10_61
			PIXEL11_30
			PIXEL20_82
			PIXEL21_32
			PIXEL22_0
			if (Diff(y[6], y[8]))
			{
				PIXEL23_0
					PIXEL32_0
					PIXEL33_0
			}
			else
			{
				PIXEL23_50
					PIXEL32_50
					PIXEL33_50
			}
		PIXEL30_82
			PIXEL31_32
			break;
	}
	case 59:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_0
				PIXEL01_0
				PIXEL10_0
		}
		else
		{
			PIXEL00_50
				PIXEL01_50
				PIXEL10_50
		}
		if (Diff(y[2], y[6]))
		{
			PIXEL02_10
				PIXEL03_80
				PIXEL12_30
				PIXEL13_10
		}
		else
		{
			PIXEL02_11
				PIXEL03_20
				PIXEL12_0
				PIXEL13_12
		}
		PIXEL11_0
			PIXEL20_31
			PIXEL21_31
			PIXEL22_30
			PIXEL23_10
			PIXEL30_81
			PIXEL31_81
			PIXEL32_61
			PIXEL33_80
			break;
	}
	case 121:
	{
		PIXEL00_82
			PIXEL01_82
			PIXEL02_61
			PIXEL03_80
			PIXEL10_32
			PIXEL11_32
			PIXEL12_30
			PIXEL13_10
			if (Diff(y[8], y[4]))
			{
				PIXEL20_0
					PIXEL30_0
					PIXEL31_0
			}
			else
			{
				PIXEL20_50
					PIXEL30_50
					PIXEL31_50
			}
		PIXEL21_0
			if (Diff(y[6], y[8]))
			{
				PIXEL22_30
					PIXEL23_10
					PIXEL32_10
					PIXEL33_80
			}
			else
			{
				PIXEL22_0
					PIXEL23_11
					PIXEL32_12
					PIXEL33_20
			}
		break;
	}
	case 87:
	{
		PIXEL00_81
			PIXEL01_31
			if (Diff(y[2], y[6]))
			{
				PIXEL02_0
					PIXEL03_0
					PIXEL13_0
			}
			else
			{
				PIXEL02_50
					PIXEL03_50
					PIXEL13_50
			}
		PIXEL10_81
			PIXEL11_31
			PIXEL12_0
			PIXEL20_61
			PIXEL21_30
			if (Diff(y[6], y[8]))
			{
				PIXEL22_30
					PIXEL23_10
					PIXEL32_10
					PIXEL33_80
			}
			else
			{
				PIXEL22_0
					PIXEL23_11
					PIXEL32_12
					PIXEL33_20
			}
		PIXEL30_80
			PIXEL31_10
			break;
	}
	case 79:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_0
				PIXEL01_0
				PIXEL10_0
		}
		else
		{
			PIXEL00_50
				PIXEL01_50
				PIXEL10_50
		}
		PIXEL02_32
			PIXEL03_82
			PIXEL11_0
			PIXEL12_32
			PIXEL13_82
			if (Diff(y[8], y[4]))
			{
				PIXEL20_10
					PIXEL21_30
					PIXEL30_80
					PIXEL31_10
			}
			else
			{
				PIXEL20_12
					PIXEL21_0
					PIXEL30_20
					PIXEL31_11
			}
		PIXEL22_30
			PIXEL23_61
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 122:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_80
				PIXEL01_10
				PIXEL10_10
				PIXEL11_30
		}
		else
		{
			PIXEL00_20
				PIXEL01_12
				PIXEL10_11
				PIXEL11_0
		}
		if (Diff(y[2], y[6]))
		{
			PIXEL02_10
				PIXEL03_80
				PIXEL12_30
				PIXEL13_10
		}
		else
		{
			PIXEL02_11
				PIXEL03_20
				PIXEL12_0
				PIXEL13_12
		}
		if (Diff(y[8], y[4]))
		{
			PIXEL20_0
				PIXEL30_0
				PIXEL31_0
		}
		else
		{
			PIXEL20_50
				PIXEL30_50
				PIXEL31_50
		}
		PIXEL21_0
			if (Diff(y[6], y[8]))
			{
				PIXEL22_30
					PIXEL23_10
					PIXEL32_10
					PIXEL33_80
			}
			else
			{
				PIXEL22_0
					PIXEL23_11
					PIXEL32_12
					PIXEL33_20
			}
		break;
	}
	case 94:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_80
				PIXEL01_10
				PIXEL10_10
				PIXEL11_30
		}
		else
		{
			PIXEL00_20
				PIXEL01_12
				PIXEL10_11
				PIXEL11_0
		}
		if (Diff(y[2], y[6]))
		{
			PIXEL02_0
				PIXEL03_0
				PIXEL13_0
		}
		else
		{
			PIXEL02_50
				PIXEL03_50
				PIXEL13_50
		}
		PIXEL12_0
			if (Diff(y[8], y[4]))
			{
				PIXEL20_10
					PIXEL21_30
					PIXEL30_80
					PIXEL31_10
			}
			else
			{
				PIXEL20_12
					PIXEL21_0
					PIXEL30_20
					PIXEL31_11
			}
		if (Diff(y[6], y[8]))
		{
			PIXEL22_30
				PIXEL23_10
				PIXEL32_10
				PIXEL33_80
		}
		else
		{
			PIXEL22_0
				PIXEL23_11
				PIXEL32_12
				PIXEL33_20
		}
		break;
	}
	case 218:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_80
				PIXEL01_10
				PIXEL10_10
				PIXEL11_30
		}
		else
		{
			PIXEL00_20
				PIXEL01_12
				PIXEL10_11
				PIXEL11_0
		}
		if (Diff(y[2], y[6]))
		{
			PIXEL02_10
				PIXEL03_80
				PIXEL12_30
				PIXEL13_10
		}
		else
		{
			PIXEL02_11
				PIXEL03_20
				PIXEL12_0
				PIXEL13_12
		}
		if (Diff(y[8], y[4]))
		{
			PIXEL20_10
				PIXEL21_30
				PIXEL30_80
				PIXEL31_10
		}
		else
		{
			PIXEL20_12
				PIXEL21_0
				PIXEL30_20
				PIXEL31_11
		}
		PIXEL22_0
			if (Diff(y[6], y[8]))
			{
				PIXEL23_0
					PIXEL32_0
					PIXEL33_0
			}
			else
			{
				PIXEL23_50
					PIXEL32_50
					PIXEL33_50
			}
		break;
	}
	case 91:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_0
				PIXEL01_0
				PIXEL10_0
		}
		else
		{
			PIXEL00_50
				PIXEL01_50
				PIXEL10_50
		}
		if (Diff(y[2], y[6]))
		{
			PIXEL02_10
				PIXEL03_80
				PIXEL12_30
				PIXEL13_10
		}
		else
		{
			PIXEL02_11
				PIXEL03_20
				PIXEL12_0
				PIXEL13_12
		}
		PIXEL11_0
			if (Diff(y[8], y[4]))
			{
				PIXEL20_10
					PIXEL21_30
					PIXEL30_80
					PIXEL31_10
			}
			else
			{
				PIXEL20_12
					PIXEL21_0
					PIXEL30_20
					PIXEL31_11
			}
		if (Diff(y[6], y[8]))
		{
			PIXEL22_30
				PIXEL23_10
				PIXEL32_10
				PIXEL33_80
		}
		else
		{
			PIXEL22_0
				PIXEL23_11
				PIXEL32_12
				PIXEL33_20
		}
		break;
	}
	case 229:
	{
		PIXEL00_20
			PIXEL01_60
			PIXEL02_60
			PIXEL03_20
			PIXEL10_60
			PIXEL11_70
			PIXEL12_70
			PIXEL13_60
			PIXEL20_82
			PIXEL21_32
			PIXEL22_31
			PIXEL23_81
			PIXEL30_82
			PIXEL31_32
			PIXEL32_31
			PIXEL33_81
			break;
	}
	case 167:
	{
		PIXEL00_81
			PIXEL01_31
			PIXEL02_32
			PIXEL03_82
			PIXEL10_81
			PIXEL11_31
			PIXEL12_32
			PIXEL13_82
			PIXEL20_60
			PIXEL21_70
			PIXEL22_70
			PIXEL23_60
			PIXEL30_20
			PIXEL31_60
			PIXEL32_60
			PIXEL33_20
			break;
	}
	case 173:
	{
		PIXEL00_82
			PIXEL01_82
			PIXEL02_60
			PIXEL03_20
			PIXEL10_32
			PIXEL11_32
			PIXEL12_70
			PIXEL13_60
			PIXEL20_31
			PIXEL21_31
			PIXEL22_70
			PIXEL23_60
			PIXEL30_81
			PIXEL31_81
			PIXEL32_60
			PIXEL33_20
			break;
	}
	case 181:
	{
		PIXEL00_20
			PIXEL01_60
			PIXEL02_81
			PIXEL03_81
			PIXEL10_60
			PIXEL11_70
			PIXEL12_31
			PIXEL13_31
			PIXEL20_60
			PIXEL21_70
			PIXEL22_32
			PIXEL23_32
			PIXEL30_20
			PIXEL31_60
			PIXEL32_82
			PIXEL33_82
			break;
	}
	case 186:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_80
				PIXEL01_10
				PIXEL10_10
				PIXEL11_30
		}
		else
		{
			PIXEL00_20
				PIXEL01_12
				PIXEL10_11
				PIXEL11_0
		}
		if (Diff(y[2], y[6]))
		{
			PIXEL02_10
				PIXEL03_80
				PIXEL12_30
				PIXEL13_10
		}
		else
		{
			PIXEL02_11
				PIXEL03_20
				PIXEL12_0
				PIXEL13_12
		}
		PIXEL20_31
			PIXEL21_31
			PIXEL22_32
			PIXEL23_32
			PIXEL30_81
			PIXEL31_81
			PIXEL32_82
			PIXEL33_82
			break;
	}
	case 115:
	{
		PIXEL00_81
			PIXEL01_31
			if (Diff(y[2], y[6]))
			{
				PIXEL02_10
					PIXEL03_80
					PIXEL12_30
					PIXEL13_10
			}
			else
			{
				PIXEL02_11
					PIXEL03_20
					PIXEL12_0
					PIXEL13_12
			}
		PIXEL10_81
			PIXEL11_31
			PIXEL20_82
			PIXEL21_32
			if (Diff(y[6], y[8]))
			{
				PIXEL22_30
					PIXEL23_10
					PIXEL32_10
					PIXEL33_80
			}
			else
			{
				PIXEL22_0
					PIXEL23_11
					PIXEL32_12
					PIXEL33_20
			}
		PIXEL30_82
			PIXEL31_32
			break;
	}
	case 93:
	{
		PIXEL00_82
			PIXEL01_82
			PIXEL02_81
			PIXEL03_81
			PIXEL10_32
			PIXEL11_32
			PIXEL12_31
			PIXEL13_31
			if (Diff(y[8], y[4]))
			{
				PIXEL20_10
					PIXEL21_30
					PIXEL30_80
					PIXEL31_10
			}
			else
			{
				PIXEL20_12
					PIXEL21_0
					PIXEL30_20
					PIXEL31_11
			}
		if (Diff(y[6], y[8]))
		{
			PIXEL22_30
				PIXEL23_10
				PIXEL32_10
				PIXEL33_80
		}
		else
		{
			PIXEL22_0
				PIXEL23_11
				PIXEL32_12
				PIXEL33_20
		}
		break;
	}
	case 206:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_80
				PIXEL01_10
				PIXEL10_10
				PIXEL11_30
		}
		else
		{
			PIXEL00_20
				PIXEL01_12
				PIXEL10_11
				PIXEL11_0
		}
		PIXEL02_32
			PIXEL03_82
			PIXEL12_32
			PIXEL13_82
			if (Diff(y[8], y[4]))
			{
				PIXEL20_10
					PIXEL21_30
					PIXEL30_80
					PIXEL31_10
			}
			else
			{
				PIXEL20_12
					PIXEL21_0
					PIXEL30_20
					PIXEL31_11
			}
		PIXEL22_31
			PIXEL23_81
			PIXEL32_31
			PIXEL33_81
			break;
	}
	case 205:
	case 201:
	{
		PIXEL00_82
			PIXEL01_82
			PIXEL02_60
			PIXEL03_20
			PIXEL10_32
			PIXEL11_32
			PIXEL12_70
			PIXEL13_60
			if (Diff(y[8], y[4]))
			{
				PIXEL20_10
					PIXEL21_30
					PIXEL30_80
					PIXEL31_10
			}
			else
			{
				PIXEL20_12
					PIXEL21_0
					PIXEL30_20
					PIXEL31_11
			}
		PIXEL22_31
			PIXEL23_81
			PIXEL32_31
			PIXEL33_81
			break;
	}
	case 174:
	case 46:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_80
				PIXEL01_10
				PIXEL10_10
				PIXEL11_30
		}
		else
		{
			PIXEL00_20
				PIXEL01_12
				PIXEL10_11
				PIXEL11_0
		}
		PIXEL02_32
			PIXEL03_82
			PIXEL12_32
			PIXEL13_82
			PIXEL20_31
			PIXEL21_31
			PIXEL22_70
			PIXEL23_60
			PIXEL30_81
			PIXEL31_81
			PIXEL32_60
			PIXEL33_20
			break;
	}
	case 179:
	case 147:
	{
		PIXEL00_81
			PIXEL01_31
			if (Diff(y[2], y[6]))
			{
				PIXEL02_10
					PIXEL03_80
					PIXEL12_30
					PIXEL13_10
			}
			else
			{
				PIXEL02_11
					PIXEL03_20
					PIXEL12_0
					PIXEL13_12
			}
		PIXEL10_81
			PIXEL11_31
			PIXEL20_60
			PIXEL21_70
			PIXEL22_32
			PIXEL23_32
			PIXEL30_20
			PIXEL31_60
			PIXEL32_82
			PIXEL33_82
			break;
	}
	case 117:
	case 116:
	{
		PIXEL00_20
			PIXEL01_60
			PIXEL02_81
			PIXEL03_81
			PIXEL10_60
			PIXEL11_70
			PIXEL12_31
			PIXEL13_31
			PIXEL20_82
			PIXEL21_32
			if (Diff(y[6], y[8]))
			{
				PIXEL22_30
					PIXEL23_10
					PIXEL32_10
					PIXEL33_80
			}
			else
			{
				PIXEL22_0
					PIXEL23_11
					PIXEL32_12
					PIXEL33_20
			}
		PIXEL30_82
			PIXEL31_32
			break;
	}
	case 189:
	{
		PIXEL00_82
			PIXEL01_82
			PIXEL02_81
			PIXEL03_81
			PIXEL10_32
			PIXEL11_32
			PIXEL12_31
			PIXEL13_31
			PIXEL20_31
			PIXEL21_31
			PIXEL22_32
			PIXEL23_32
			PIXEL30_81
			PIXEL31_81
			PIXEL32_82
			PIXEL33_82
			break;
	}
	case 231:
	{
		PIXEL00_81
			PIXEL01_31
			PIXEL02_32
			PIXEL03_82
			PIXEL10_81
			PIXEL11_31
			PIXEL12_32
			PIXEL13_82
			PIXEL20_82
			PIXEL21_32
			PIXEL22_31
			PIXEL23_81
			PIXEL30_82
			PIXEL31_32
			PIXEL32_31
			PIXEL33_81
			break;
	}
	case 126:
	{
		PIXEL00_80
			PIXEL01_10
			if (Diff(y[2], y[6]))
			{
				PIXEL02_0
					PIXEL03_0
					PIXEL13_0
			}
			else
			{
				PIXEL02_50
					PIXEL03_50
					PIXEL13_50
			}
		PIXEL10_10
			PIXEL11_30
			PIXEL12_0
			if (Diff(y[8], y[4]))
			{
				PIXEL20_0
					PIXEL30_0
					PIXEL31_0
			}
			else
			{
				PIXEL20_50
					PIXEL30_50
					PIXEL31_50
			}
		PIXEL21_0
			PIXEL22_30
			PIXEL23_10
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 219:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_0
				PIXEL01_0
				PIXEL10_0
		}
		else
		{
			PIXEL00_50
				PIXEL01_50
				PIXEL10_50
		}
		PIXEL02_10
			PIXEL03_80
			PIXEL11_0
			PIXEL12_30
			PIXEL13_10
			PIXEL20_10
			PIXEL21_30
			PIXEL22_0
			if (Diff(y[6], y[8]))
			{
				PIXEL23_0
					PIXEL32_0
					PIXEL33_0
			}
			else
			{
				PIXEL23_50
					PIXEL32_50
					PIXEL33_50
			}
		PIXEL30_80
			PIXEL31_10
			break;
	}
	case 125:
	{
		if (Diff(y[8], y[4]))
		{
			PIXEL00_82
				PIXEL10_32
				PIXEL20_0
				PIXEL21_0
				PIXEL30_0
				PIXEL31_0
		}
		else
		{
			PIXEL00_11
				PIXEL10_13
				PIXEL20_83
				PIXEL21_70
				PIXEL30_50
				PIXEL31_21
		}
		PIXEL01_82
			PIXEL02_81
			PIXEL03_81
			PIXEL11_32
			PIXEL12_31
			PIXEL13_31
			PIXEL22_30
			PIXEL23_10
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 221:
	{
		PIXEL00_82
			PIXEL01_82
			PIXEL02_81
			if (Diff(y[6], y[8]))
			{
				PIXEL03_81
					PIXEL13_31
					PIXEL22_0
					PIXEL23_0
					PIXEL32_0
					PIXEL33_0
			}
			else
			{
				PIXEL03_12
					PIXEL13_14
					PIXEL22_70
					PIXEL23_83
					PIXEL32_21
					PIXEL33_50
			}
		PIXEL10_32
			PIXEL11_32
			PIXEL12_31
			PIXEL20_10
			PIXEL21_30
			PIXEL30_80
			PIXEL31_10
			break;
	}
	case 207:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_0
				PIXEL01_0
				PIXEL02_32
				PIXEL03_82
				PIXEL10_0
				PIXEL11_0
		}
		else
		{
			PIXEL00_50
				PIXEL01_83
				PIXEL02_13
				PIXEL03_11
				PIXEL10_21
				PIXEL11_70
		}
		PIXEL12_32
			PIXEL13_82
			PIXEL20_10
			PIXEL21_30
			PIXEL22_31
			PIXEL23_81
			PIXEL30_80
			PIXEL31_10
			PIXEL32_31
			PIXEL33_81
			break;
	}
	case 238:
	{
		PIXEL00_80
			PIXEL01_10
			PIXEL02_32
			PIXEL03_82
			PIXEL10_10
			PIXEL11_30
			PIXEL12_32
			PIXEL13_82
			if (Diff(y[8], y[4]))
			{
				PIXEL20_0
					PIXEL21_0
					PIXEL30_0
					PIXEL31_0
					PIXEL32_31
					PIXEL33_81
			}
			else
			{
				PIXEL20_21
					PIXEL21_70
					PIXEL30_50
					PIXEL31_83
					PIXEL32_14
					PIXEL33_12
			}
		PIXEL22_31
			PIXEL23_81
			break;
	}
	case 190:
	{
		PIXEL00_80
			PIXEL01_10
			if (Diff(y[2], y[6]))
			{
				PIXEL02_0
					PIXEL03_0
					PIXEL12_0
					PIXEL13_0
					PIXEL23_32
					PIXEL33_82
			}
			else
			{
				PIXEL02_21
					PIXEL03_50
					PIXEL12_70
					PIXEL13_83
					PIXEL23_13
					PIXEL33_11
			}
		PIXEL10_10
			PIXEL11_30
			PIXEL20_31
			PIXEL21_31
			PIXEL22_32
			PIXEL30_81
			PIXEL31_81
			PIXEL32_82
			break;
	}
	case 187:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_0
				PIXEL01_0
				PIXEL10_0
				PIXEL11_0
				PIXEL20_31
				PIXEL30_81
		}
		else
		{
			PIXEL00_50
				PIXEL01_21
				PIXEL10_83
				PIXEL11_70
				PIXEL20_14
				PIXEL30_12
		}
		PIXEL02_10
			PIXEL03_80
			PIXEL12_30
			PIXEL13_10
			PIXEL21_31
			PIXEL22_32
			PIXEL23_32
			PIXEL31_81
			PIXEL32_82
			PIXEL33_82
			break;
	}
	case 243:
	{
		PIXEL00_81
			PIXEL01_31
			PIXEL02_10
			PIXEL03_80
			PIXEL10_81
			PIXEL11_31
			PIXEL12_30
			PIXEL13_10
			PIXEL20_82
			PIXEL21_32
			if (Diff(y[6], y[8]))
			{
				PIXEL22_0
					PIXEL23_0
					PIXEL30_82
					PIXEL31_32
					PIXEL32_0
					PIXEL33_0
			}
			else
			{
				PIXEL22_70
					PIXEL23_21
					PIXEL30_11
					PIXEL31_13
					PIXEL32_83
					PIXEL33_50
			}
		break;
	}
	case 119:
	{
		if (Diff(y[2], y[6]))
		{
			PIXEL00_81
				PIXEL01_31
				PIXEL02_0
				PIXEL03_0
				PIXEL12_0
				PIXEL13_0
		}
		else
		{
			PIXEL00_12
				PIXEL01_14
				PIXEL02_83
				PIXEL03_50
				PIXEL12_70
				PIXEL13_21
		}
		PIXEL10_81
			PIXEL11_31
			PIXEL20_82
			PIXEL21_32
			PIXEL22_30
			PIXEL23_10
			PIXEL30_82
			PIXEL31_32
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 237:
	case 233:
	{
		PIXEL00_82
			PIXEL01_82
			PIXEL02_60
			PIXEL03_20
			PIXEL10_32
			PIXEL11_32
			PIXEL12_70
			PIXEL13_60
			PIXEL20_0
			PIXEL21_0
			PIXEL22_31
			PIXEL23_81
			if (Diff(y[8], y[4]))
			{
				PIXEL30_0
			}
			else
			{
				PIXEL30_20
			}
		PIXEL31_0
			PIXEL32_31
			PIXEL33_81
			break;
	}
	case 175:
	case 47:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_0
		}
		else
		{
			PIXEL00_20
		}
		PIXEL01_0
			PIXEL02_32
			PIXEL03_82
			PIXEL10_0
			PIXEL11_0
			PIXEL12_32
			PIXEL13_82
			PIXEL20_31
			PIXEL21_31
			PIXEL22_70
			PIXEL23_60
			PIXEL30_81
			PIXEL31_81
			PIXEL32_60
			PIXEL33_20
			break;
	}
	case 183:
	case 151:
	{
		PIXEL00_81
			PIXEL01_31
			PIXEL02_0
			if (Diff(y[2], y[6]))
			{
				PIXEL03_0
			}
			else
			{
				PIXEL03_20
			}
		PIXEL10_81
			PIXEL11_31
			PIXEL12_0
			PIXEL13_0
			PIXEL20_60
			PIXEL21_70
			PIXEL22_32
			PIXEL23_32
			PIXEL30_20
			PIXEL31_60
			PIXEL32_82
			PIXEL33_82
			break;
	}
	case 245:
	case 244:
	{
		PIXEL00_20
			PIXEL01_60
			PIXEL02_81
			PIXEL03_81
			PIXEL10_60
			PIXEL11_70
			PIXEL12_31
			PIXEL13_31
			PIXEL20_82
			PIXEL21_32
			PIXEL22_0
			PIXEL23_0
			PIXEL30_82
			PIXEL31_32
			PIXEL32_0
			if (Diff(y[6], y[8]))
			{
				PIXEL33_0
			}
			else
			{
				PIXEL33_20
			}
		break;
	}
	case 250:
	{
		PIXEL00_80
			PIXEL01_10
			PIXEL02_10
			PIXEL03_80
			PIXEL10_10
			PIXEL11_30
			PIXEL12_30
			PIXEL13_10
			if (Diff(y[8], y[4]))
			{
				PIXEL20_0
					PIXEL30_0
					PIXEL31_0
			}
			else
			{
				PIXEL20_50
					PIXEL30_50
					PIXEL31_50
			}
		PIXEL21_0
			PIXEL22_0
			if (Diff(y[6], y[8]))
			{
				PIXEL23_0
					PIXEL32_0
					PIXEL33_0
			}
			else
			{
				PIXEL23_50
					PIXEL32_50
					PIXEL33_50
			}
		break;
	}
	case 123:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_0
				PIXEL01_0
				PIXEL10_0
		}
		else
		{
			PIXEL00_50
				PIXEL01_50
				PIXEL10_50
		}
		PIXEL02_10
			PIXEL03_80
			PIXEL11_0
			PIXEL12_30
			PIXEL13_10
			if (Diff(y[8], y[4]))
			{
				PIXEL20_0
					PIXEL30_0
					PIXEL31_0
			}
			else
			{
				PIXEL20_50
					PIXEL30_50
					PIXEL31_50
			}
		PIXEL21_0
			PIXEL22_30
			PIXEL23_10
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 95:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_0
				PIXEL01_0
				PIXEL10_0
		}
		else
		{
			PIXEL00_50
				PIXEL01_50
				PIXEL10_50
		}
		if (Diff(y[2], y[6]))
		{
			PIXEL02_0
				PIXEL03_0
				PIXEL13_0
		}
		else
		{
			PIXEL02_50
				PIXEL03_50
				PIXEL13_50
		}
		PIXEL11_0
			PIXEL12_0
			PIXEL20_10
			PIXEL21_30
			PIXEL22_30
			PIXEL23_10
			PIXEL30_80
			PIXEL31_10
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 222:
	{
		PIXEL00_80
			PIXEL01_10
			if (Diff(y[2], y[6]))
			{
				PIXEL02_0
					PIXEL03_0
					PIXEL13_0
			}
			else
			{
				PIXEL02_50
					PIXEL03_50
					PIXEL13_50
			}
		PIXEL10_10
			PIXEL11_30
			PIXEL12_0
			PIXEL20_10
			PIXEL21_30
			PIXEL22_0
			if (Diff(y[6], y[8]))
			{
				PIXEL23_0
					PIXEL32_0
					PIXEL33_0
			}
			else
			{
				PIXEL23_50
					PIXEL32_50
					PIXEL33_50
			}
		PIXEL30_80
			PIXEL31_10
			break;
	}
	case 252:
	{
		PIXEL00_80
			PIXEL01_61
			PIXEL02_81
			PIXEL03_81
			PIXEL10_10
			PIXEL11_30
			PIXEL12_31
			PIXEL13_31
			if (Diff(y[8], y[4]))
			{
				PIXEL20_0
					PIXEL30_0
					PIXEL31_0
			}
			else
			{
				PIXEL20_50
					PIXEL30_50
					PIXEL31_50
			}
		PIXEL21_0
			PIXEL22_0
			PIXEL23_0
			PIXEL32_0
			if (Diff(y[6], y[8]))
			{
				PIXEL33_0
			}
			else
			{
				PIXEL33_20
			}
		break;
	}
	case 249:
	{
		PIXEL00_82
			PIXEL01_82
			PIXEL02_61
			PIXEL03_80
			PIXEL10_32
			PIXEL11_32
			PIXEL12_30
			PIXEL13_10
			PIXEL20_0
			PIXEL21_0
			PIXEL22_0
			if (Diff(y[6], y[8]))
			{
				PIXEL23_0
					PIXEL32_0
					PIXEL33_0
			}
			else
			{
				PIXEL23_50
					PIXEL32_50
					PIXEL33_50
			}
		if (Diff(y[8], y[4]))
		{
			PIXEL30_0
		}
		else
		{
			PIXEL30_20
		}
		PIXEL31_0
			break;
	}
	case 235:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_0
				PIXEL01_0
				PIXEL10_0
		}
		else
		{
			PIXEL00_50
				PIXEL01_50
				PIXEL10_50
		}
		PIXEL02_10
			PIXEL03_80
			PIXEL11_0
			PIXEL12_30
			PIXEL13_61
			PIXEL20_0
			PIXEL21_0
			PIXEL22_31
			PIXEL23_81
			if (Diff(y[8], y[4]))
			{
				PIXEL30_0
			}
			else
			{
				PIXEL30_20
			}
		PIXEL31_0
			PIXEL32_31
			PIXEL33_81
			break;
	}
	case 111:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_0
		}
		else
		{
			PIXEL00_20
		}
		PIXEL01_0
			PIXEL02_32
			PIXEL03_82
			PIXEL10_0
			PIXEL11_0
			PIXEL12_32
			PIXEL13_82
			if (Diff(y[8], y[4]))
			{
				PIXEL20_0
					PIXEL30_0
					PIXEL31_0
			}
			else
			{
				PIXEL20_50
					PIXEL30_50
					PIXEL31_50
			}
		PIXEL21_0
			PIXEL22_30
			PIXEL23_61
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 63:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_0
		}
		else
		{
			PIXEL00_20
		}
		PIXEL01_0
			if (Diff(y[2], y[6]))
			{
				PIXEL02_0
					PIXEL03_0
					PIXEL13_0
			}
			else
			{
				PIXEL02_50
					PIXEL03_50
					PIXEL13_50
			}
		PIXEL10_0
			PIXEL11_0
			PIXEL12_0
			PIXEL20_31
			PIXEL21_31
			PIXEL22_30
			PIXEL23_10
			PIXEL30_81
			PIXEL31_81
			PIXEL32_61
			PIXEL33_80
			break;
	}
	case 159:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_0
				PIXEL01_0
				PIXEL10_0
		}
		else
		{
			PIXEL00_50
				PIXEL01_50
				PIXEL10_50
		}
		PIXEL02_0
			if (Diff(y[2], y[6]))
			{
				PIXEL03_0
			}
			else
			{
				PIXEL03_20
			}
		PIXEL11_0
			PIXEL12_0
			PIXEL13_0
			PIXEL20_10
			PIXEL21_30
			PIXEL22_32
			PIXEL23_32
			PIXEL30_80
			PIXEL31_61
			PIXEL32_82
			PIXEL33_82
			break;
	}
	case 215:
	{
		PIXEL00_81
			PIXEL01_31
			PIXEL02_0
			if (Diff(y[2], y[6]))
			{
				PIXEL03_0
			}
			else
			{
				PIXEL03_20
			}
		PIXEL10_81
			PIXEL11_31
			PIXEL12_0
			PIXEL13_0
			PIXEL20_61
			PIXEL21_30
			PIXEL22_0
			if (Diff(y[6], y[8]))
			{
				PIXEL23_0
					PIXEL32_0
					PIXEL33_0
			}
			else
			{
				PIXEL23_50
					PIXEL32_50
					PIXEL33_50
			}
		PIXEL30_80
			PIXEL31_10
			break;
	}
	case 246:
	{
		PIXEL00_80
			PIXEL01_10
			if (Diff(y[2], y[6]))
			{
				PIXEL02_0
					PIXEL03_0
					PIXEL13_0
			}
			else
			{
				PIXEL02_50
					PIXEL03_50
					PIXEL13_50
			}
		PIXEL10_61
			PIXEL11_30
			PIXEL12_0
			PIXEL20_82
			PIXEL21_32
			PIXEL22_0
			PIXEL23_0
			PIXEL30_82
			PIXEL31_32
			PIXEL32_0
			if (Diff(y[6], y[8]))
			{
				PIXEL33_0
			}
			else
			{
				PIXEL33_20
			}
		break;
	}
	case 254:
	{
		PIXEL00_80
			PIXEL01_10
			if (Diff(y[2], y[6]))
			{
				PIXEL02_0
					PIXEL03_0
					PIXEL13_0
			}
			else
			{
				PIXEL02_50
					PIXEL03_50
					PIXEL13_50
			}
		PIXEL10_10
			PIXEL11_30
			PIXEL12_0
			if (Diff(y[8], y[4]))
			{
				PIXEL20_0
					PIXEL30_0
					PIXEL31_0
			}
			else
			{
				PIXEL20_50
					PIXEL30_50
					PIXEL31_50
			}
		PIXEL21_0
			PIXEL22_0
			PIXEL23_0
			PIXEL32_0
			if (Diff(y[6], y[8]))
			{
				PIXEL33_0
			}
			else
			{
				PIXEL33_20
			}
		break;
	}
	case 253:
	{
		PIXEL00_82
			PIXEL01_82
			PIXEL02_81
			PIXEL03_81
			PIXEL10_32
			PIXEL11_32
			PIXEL12_31
			PIXEL13_31
			PIXEL20_0
			PIXEL21_0
			PIXEL22_0
			PIXEL23_0
			if (Diff(y[8], y[4]))
			{
				PIXEL30_0
			}
			else
			{
				PIXEL30_20
			}
		PIXEL31_0
			PIXEL32_0
			if (Diff(y[6], y[8]))
			{
				PIXEL33_0
			}
			else
			{
				PIXEL33_20
			}
		break;
	}
	case 251:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_0
				PIXEL01_0
				PIXEL10_0
		}
		else
		{
			PIXEL00_50
				PIXEL01_50
				PIXEL10_50
		}
		PIXEL02_10
			PIXEL03_80
			PIXEL11_0
			PIXEL12_30
			PIXEL13_10
			PIXEL20_0
			PIXEL21_0
			PIXEL22_0
			if (Diff(y[6], y[8]))
			{
				PIXEL23_0
					PIXEL32_0
					PIXEL33_0
			}
			else
			{
				PIXEL23_50
					PIXEL32_50
					PIXEL33_50
			}
		if (Diff(y[8], y[4]))
		{
			PIXEL30_0
		}
		else
		{
			PIXEL30_20
		}
		PIXEL31_0
			break;
	}
	case 239:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_0
		}
		else
		{
			PIXEL00_20
		}
		PIXEL01_0
			PIXEL02_32
			PIXEL03_82
			PIXEL10_0
			PIXEL11_0
			PIXEL12_32
			PIXEL13_82
			PIXEL20_0
			PIXEL21_0
			PIXEL22_31
			PIXEL23_81
			if (Diff(y[8], y[4]))
			{
				PIXEL30_0
			}
			else
			{
				PIXEL30_20
			}
		PIXEL31_0
			PIXEL32_31
			PIXEL33_81
			break;
	}
	case 127:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_0
		}
		else
		{
			PIXEL00_20
		}
		PIXEL01_0
			if (Diff(y[2], y[6]))
			{
				PIXEL02_0
					PIXEL03_0
					PIXEL13_0
			}
			else
			{
				PIXEL02_50
					PIXEL03_50
					PIXEL13_50
			}
		PIXEL10_0
			PIXEL11_0
			PIXEL12_0
			if (Diff(y[8], y[4]))
			{
				PIXEL20_0
					PIXEL30_0
					PIXEL31_0
			}
			else
			{
				PIXEL20_50
					PIXEL30_50
					PIXEL31_50
			}
		PIXEL21_0
			PIXEL22_30
			PIXEL23_10
			PIXEL32_10
			PIXEL33_80
			break;
	}
	case 191:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_0
		}
		else
		{
			PIXEL00_20
		}
		PIXEL01_0
			PIXEL02_0
			if (Diff(y[2], y[6]))
			{
				PIXEL03_0
			}
			else
			{
				PIXEL03_20
			}
		PIXEL10_0
			PIXEL11_0
			PIXEL12_0
			PIXEL13_0
			PIXEL20_31
			PIXEL21_31
			PIXEL22_32
			PIXEL23_32
			PIXEL30_81
			PIXEL31_81
			PIXEL32_82
			PIXEL33_82
			break;
	}
	case 223:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_0
				PIXEL01_0
				PIXEL10_0
		}
		else
		{
			PIXEL00_50
				PIXEL01_50
				PIXEL10_50
		}
		PIXEL02_0
			if (Diff(y[2], y[6]))
			{
				PIXEL03_0
			}
			else
			{
				PIXEL03_20
			}
		PIXEL11_0
			PIXEL12_0
			PIXEL13_0
			PIXEL20_10
			PIXEL21_30
			PIXEL22_0
			if (Diff(y[6], y[8]))
			{
				PIXEL23_0
					PIXEL32_0
					PIXEL33_0
			}
			else
			{
				PIXEL23_50
					PIXEL32_50
					PIXEL33_50
			}
		PIXEL30_80
			PIXEL31_10
			break;
	}
	case 247:
	{
		PIXEL00_81
			PIXEL01_31
			PIXEL02_0
			if (Diff(y[2], y[6]))
			{
				PIXEL03_0
			}
			else
			{
				PIXEL03_20
			}
		PIXEL10_81
			PIXEL11_31
			PIXEL12_0
			PIXEL13_0
			PIXEL20_82
			PIXEL21_32
			PIXEL22_0
			PIXEL23_0
			PIXEL30_82
			PIXEL31_32
			PIXEL32_0
			if (Diff(y[6], y[8]))
			{
				PIXEL33_0
			}
			else
			{
				PIXEL33_20
			}
		break;
	}
	case 255:
	{
		if (Diff(y[4], y[2]))
		{
			PIXEL00_0
		}
		else
		{
			PIXEL00_20
		}
		PIXEL01_0
			PIXEL02_0
			if (Diff(y[2], y[6]))
			{
				PIXEL03_0
			}
			else
			{
				PIXEL03_20
			}
		PIXEL10_0
			PIXEL11_0
			PIXEL12_0
			PIXEL13_0
			PIXEL20_0
			PIXEL21_0
			PIXEL22_0
			PIXEL23_0
			if (Diff(y[8], y[4]))
			{
				PIXEL30_0
			}
			else
			{
				PIXEL30_20
			}
		PIXEL31_0
			PIXEL32_0
			if (Diff(y[6], y[8]))
			{
				PIXEL33_0
			}
			else
			{
				PIXEL33_20
			}
		break;
	}
	}
}