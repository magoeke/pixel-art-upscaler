#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdint.h>
#include <stdlib.h>
#include <IL/il.h>
#include <stdint.h>
#include "common.cuh"
#include "pixel_functions.cuh"
#include "image.h"
#include <cstdio>
#include <iostream>
#include "gputimer.h"

#define FACTOR 4

#include <stdio.h>

uint32_t   RGBtoYUV[16777216];

void hqxInit();
void hq4x_32(uint32_t * sp, uint32_t * dp, int Xres, int Yres, uint32_t*, uint32_t*);
__global__ void hq4x(uint32_t*, uint32_t*, int, int, uint32_t*, FunctionPointer*);
__global__ void ConvertRGBtoYUV(uint32_t*, uint32_t*, uint32_t*, int);

int main()
{
	ILuint handle;
	ilInit();
	ilEnable(IL_ORIGIN_SET);
	ilGenImages(1, &handle);
	ilBindImage(handle);
	hipError_t cudaStatus;
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);

	std::cout << "Read images" << std::endl;
	Image *org = new Image("image.png");
	Image *orgy = new Image("image.png");
	uint32_t *input, *out, *yuv;
	
	// create RGBtoYUV table
	hqxInit();
	
	// calculated yuv values for all pixels in the input image
	hipMalloc(&input, org->getHeight() * org->getWidth() * sizeof(uint32_t));
	hipMalloc(&out, org->getHeight() * org->getWidth() * sizeof(uint32_t));
	hipMalloc(&yuv, 16777216 * sizeof(uint32_t));

	hipMemcpy(input, org->getData(), org->getHeight() * org->getWidth() * sizeof(uint32_t), hipMemcpyHostToDevice);
	hipMemcpy(yuv, RGBtoYUV, 16777216 * sizeof(uint32_t), hipMemcpyHostToDevice);
	
	ConvertRGBtoYUV <<<((org->getHeight() * org->getWidth()) / deviceProp.maxThreadsPerBlock)+1, deviceProp.maxThreadsPerBlock >>>(input, out, yuv, deviceProp.maxThreadsPerBlock);
	
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	}

	// save yuv image to image structure
	cudaStatus = hipMemcpy(orgy->getData(), out, orgy->getHeight() * orgy->getWidth() * sizeof(uint32_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
	hipFree(yuv);
	Image *res = new Image("image2.png", org->getWidth() * FACTOR, org->getHeight() * FACTOR);
	
	hq4x_32(org->getData(), res->getData(), org->getWidth(), org->getHeight(), input, out);
	
	std::cout << "Save new image" << std::endl;
	res->saveImage();
	
	std::cout << "Free space" << std::endl;
	delete org;
	delete res;
	ilDeleteImages(1, &handle);
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
    
	return 0;
}

__global__ void ConvertRGBtoYUV(uint32_t *sp, uint32_t *br, uint32_t *array, int maxThreads) {
	int i = threadIdx.x + maxThreads * blockIdx.x;
	br[i] = rgb_to_yuv(sp[i],array);
}

void hqxInit(void)
{
	/* Initalize RGB to YUV lookup table */
	uint32_t c, r, g, b, y, u, v;
	for (c = 0; c < 16777215; c++) {
		r = (c & 0xFF0000) >> 16;
		g = (c & 0x00FF00) >> 8;
		b = c & 0x0000FF;
		y = (uint32_t)(0.299*r + 0.587*g + 0.114*b);
		u = (uint32_t)(-0.169*r - 0.331*g + 0.5*b) + 128;
		v = (uint32_t)(0.5*r - 0.419*g - 0.081*b) + 128;
		RGBtoYUV[c] = (y << 16) + (u << 8) + v;
	}
}

void hq4x_32(uint32_t * sp, uint32_t * dp, int Xres, int Yres, uint32_t *input, uint32_t *yuv)
{
	hipError_t cudaStatus;
	uint32_t *out;
	FunctionPointer *fpo;
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	cudaStatus = hipSetDevice(0);
	hipMalloc(&out, Xres * FACTOR * Yres * FACTOR * sizeof(uint32_t));
	hipMalloc(&fpo, 256 * sizeof(FunctionPointer));
	
	// initialize kernelfunction in device memory
	initFunctions <<<1, 1>>> (fpo);
	cudaStatus = hipDeviceSynchronize();

	int blockdimension = (Xres*Yres) > deviceProp.maxThreadsPerBlock ? (Xres*Yres) / deviceProp.maxThreadsPerBlock : 1;
	
	GpuTimer timer;
	timer.Start();
	hq4x<<<blockdimension, deviceProp.maxThreadsPerBlock>>>(input, out, Xres, Yres, yuv, fpo);
	
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}
	cudaStatus = hipDeviceSynchronize();
	timer.Stop();
	std::cout << "Timer: " << timer.Elapsed() << std::endl;
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	}

	// copy result to image
	cudaStatus = hipMemcpy(dp, out,Xres *FACTOR * Yres * FACTOR * sizeof(uint32_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	hipFree(input);
	hipFree(out);
	hipFree(yuv);
	hipFree(fpo);
}

__global__ void hq4x(uint32_t * sp, uint32_t * dp, int Xres, int Yres, uint32_t *yuv, FunctionPointer *functions)
{
	int  k;
	int  prevline, nextline;
	uint32_t w[10];
	uint32_t y[10];
	uint32_t yuv1, yuv2;
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	int row = index / Xres;
	int col = index % Xres;
	int dpL = Xres*FACTOR;
	int spL = Xres;
	if (index < Xres) prevline = 0; else prevline = index - spL;
	if (index > Xres*(Yres - 1)) nextline = 0; else nextline = index + spL;
	int indexVier = (row * Xres * FACTOR + col) * FACTOR;
	w[2] = sp[prevline];
	w[5] = sp[index];
	w[8] = sp[nextline];
	y[2] = yuv[prevline];
	y[5] = yuv[index];
	y[8] = yuv[nextline];
	if ((index%Xres)>0)
	{
		w[1] = sp[prevline - 1];
		w[4] = sp[index - 1];
		w[7] = sp[nextline - 1];
		y[1] = yuv[prevline - 1];
		y[4] = yuv[index - 1];
		y[7] = yuv[nextline - 1];
	}
	else
	{
		w[1] = w[2];
		w[4] = w[5];
		w[7] = w[8];
		y[1] = y[2];
		y[4] = y[5];
		y[7] = y[8];
	}
	if ((index%Xres)<Xres - 1)
	{
		w[3] = sp[prevline + 1];
		w[6] = sp[index + 1];
		w[9] = sp[nextline + 1];
		y[3] = yuv[prevline + 1];
		y[6] = yuv[index + 1];
		y[9] = yuv[nextline + 1];
	}
	else
	{
		w[3] = w[2];
		w[6] = w[5];
		w[9] = w[8];
		y[3] = y[2];
		y[6] = y[5];
		y[9] = y[8];
	}
	int pattern = 0;
	int flag = 1;
	yuv1 = y[5];
	for (k = 1; k <= 9; k++)
	{
		if (k == 5) continue;
		if (w[k] != w[5])
		{
			yuv2 = y[k];
			if (yuv_diff(yuv1, yuv2))
				pattern |= flag;
		}
		flag <<= 1;
	}
	functions[pattern](dp, indexVier, dpL, w, y);
	
}